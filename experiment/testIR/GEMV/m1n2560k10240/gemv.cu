#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <math.h>

// cal offset from row col and ld , in row-major matrix, ld is the width of the matrix
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

// transfer float4
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])

#define checkCudaErrors(func)                                                      \
    {                                                                              \
        hipError_t e = (func);                                                    \
        if (e != hipSuccess)                                                      \
            printf("%s %d CUDA: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
    }

int main(int argc, char **argv)
{
    if (argc != 3)
    {
        printf("usage: ./main [M] [N]\n");
        exit(0);
    }
    size_t M = atoi(argv[1]);
    size_t N = atoi(argv[2]);

    size_t bytes_A = sizeof(float) * M * N;
    size_t bytes_x = sizeof(float) * N;
    size_t bytes_y = sizeof(float) * M;
    float *h_A = (float *)malloc(bytes_A);
    float *h_x = (float *)malloc(bytes_x);
    float *h_y = (float *)malloc(bytes_y);
    float *h_y1 = (float *)malloc(bytes_y);

    float *d_A;
    float *d_x;
    float *d_y;

    checkCudaErrors(hipMalloc(&d_A, bytes_A));
    checkCudaErrors(hipMalloc(&d_x, bytes_x));
    checkCudaErrors(hipMalloc(&d_y, bytes_y));

    // 生成A的数据
    for (int i = 0; i < M * N; i++)
    {
        h_A[i] = (float)i / N;
    }

    // 生成x的数据
    for (int i = 0; i < N; i++)
    {
        h_x[i] = 1;
    }
    memset(h_y, 0, M * sizeof(float));
    memset(h_y1, 0, M * sizeof(float));

    int nIter = 1000;
    checkCudaErrors(hipMemcpy(d_A, h_A, bytes_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_x, h_x, bytes_x, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_y, h_y, bytes_y, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(h_y, d_y, bytes_y, hipMemcpyDeviceToHost));

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    // cublas
    hipblasHandle_t blas_handle;
    hipblasCreate(&blas_handle);
    float alpha = 1.0;
    float beta = 0;
    checkCudaErrors(hipMemcpy(d_y, h_y1, bytes_y, hipMemcpyHostToDevice));

    checkCudaErrors(hipEventRecord(start, 0));
    hipblasSgemv(blas_handle, HIPBLAS_OP_T,
                N, M, &alpha,
                d_A, N, d_x, 1, &beta, d_y, 1);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));

    float gpu_time = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));
    printf("GPU execution time for cublasSgemv: %.3f ms\n", gpu_time);
    checkCudaErrors(hipMemcpy(h_y1, d_y, bytes_y, hipMemcpyDeviceToHost));
    hipblasDestroy(blas_handle);

    // Free Memory
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);

    free(h_A);
    free(h_x);
    free(h_y);
    free(h_y1);
}
