#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <math.h>
#include <mma.h>
//for int8
#include <sm_61_intrinsics.h>

#include <hip/hip_fp16.h>
__device__ half max(half a, half b)
{
  return __hgt(__half(a), __half(b)) ? a : b;
}
__device__ half min(half a, half b)
{
  return __hlt(__half(a), __half(b)) ? a : b;
}

#define __int8_t_defined

#define CUDA_UNSUPPORTED_HALF_MATH_BINARY(HALF_MATH_NAME, FP32_MATH_NAME) static inline __device__ __host__ half HALF_MATH_NAME(half x, half y) {     float tmp_x = __half2float(x);                                            float tmp_y = __half2float(y);                                            float result = FP32_MATH_NAME(tmp_x, tmp_y);                              return __float2half(result);                                            }

#define CUDA_UNSUPPORTED_HALF_MATH_UNARY(HALF_MATH_NAME, FP32_MATH_NAME) static inline __device__ __host__ half HALF_MATH_NAME(half x) {            float tmp_x = __half2float(x);                                           float result = FP32_MATH_NAME(tmp_x);                                    return __float2half(result);                                           }

CUDA_UNSUPPORTED_HALF_MATH_BINARY(hpow, powf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(htanh, tanhf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(htan, tanf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(hatan, atanf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(herf, erf)

#undef CUDA_UNSUPPORTED_HALF_MATH_BINARY
#undef CUDA_UNSUPPORTED_HALF_MATH_UNARY

// Pack two half values.
static inline __device__ __host__ unsigned
__pack_half2(const half x, const half y) {
  unsigned v0 = *((unsigned short *)&x);
  unsigned v1 = *((unsigned short *)&y);
  return (v1 << 16) | v0;
}

// There is no make_int8 in cuda, but TVM codegen seem to use it
static inline __device__ longlong4 make_int8(int x0, int x1, int x2, int x3, int x4, int x5, int x6, int x7) {
  int2 i0 = make_int2(x0, x1);
  int2 i1 = make_int2(x2, x3);
  int2 i2 = make_int2(x4, x5);
  int2 i3 = make_int2(x6, x7);
  long long l0 = *(long long*)&i0;
  long long l1 = *(long long*)&i1;
  long long l2 = *(long long*)&i2;
  long long l3 = *(long long*)&i3;
  return make_longlong4(l0, l1, l2, l3);
}


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) ||      (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(1024) fused_softmax_cast_n_1_to_1024__kernel(float* __restrict__ A, half* __restrict__ compute_intermediate, int64_t n);
extern "C" __global__ void __launch_bounds__(128) fused_softmax_cast_n_1025_to_2048__kernel(float* __restrict__ A, half* __restrict__ compute_intermediate, int64_t n);
extern "C" __global__ void __launch_bounds__(160) fused_softmax_cast_n_2049_to_2560__kernel(float* __restrict__ A, half* __restrict__ compute_intermediate, int64_t n);
extern "C" __global__ void __launch_bounds__(192) fused_softmax_cast_n_2561_to_3072__kernel(float* __restrict__ A, half* __restrict__ compute_intermediate, int64_t n);
extern "C" __global__ void __launch_bounds__(192) fused_softmax_cast_n_3073_to_3584__kernel(float* __restrict__ A, half* __restrict__ compute_intermediate, int64_t n);
extern "C" __global__ void __launch_bounds__(256) fused_softmax_cast_n_3585_to_4096__kernel(float* __restrict__ A, half* __restrict__ compute_intermediate, int64_t n);
extern "C" __global__ void __launch_bounds__(1024) fused_softmax_cast_n_1_to_1024__kernel(float* __restrict__ A, half* __restrict__ compute_intermediate, int64_t n) {
  float in_thread_T_softmax_maxelem_shared[1];
  __shared__ float red_result[1];
  __shared__ float T_softmax_maxelem_shared[1];
  float in_thread_T_softmax_expsum_shared[1];
  __shared__ float red_result_1[1];
  __shared__ float T_softmax_expsum_shared[1];
  in_thread_T_softmax_maxelem_shared[0] = -3.402823e+38f;
  for (int64_t ax1_fused_0 = 0; ax1_fused_0 < ((n + (int64_t)1023) >> (int64_t)10); ++ax1_fused_0) {
    if (((ax1_fused_0 * (int64_t)1024) + ((int64_t)threadIdx.x)) < n) {
      in_thread_T_softmax_maxelem_shared[0] = max(in_thread_T_softmax_maxelem_shared[0], A[(((ax1_fused_0 * (int64_t)1024) + (((int64_t)blockIdx.x) * n)) + ((int64_t)threadIdx.x))]);
    }
  }
  float red_buf0[1];
  uint mask[1];
  float t0[1];
  float red_buf0_1[1];
  uint mask_1[1];
  float t0_1[1];
  __shared__ float red_buf_staging[32];
  red_buf0_1[0] = in_thread_T_softmax_maxelem_shared[0];
  mask_1[0] = __activemask();
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 16, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 8, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 4, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 2, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 1, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  if ((((int64_t)threadIdx.x) % (int64_t)32) == (int64_t)0) {
    red_buf_staging[(((int64_t)threadIdx.x) >> (int64_t)5)] = red_buf0_1[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) < (int64_t)32) {
    red_buf0[0] = red_buf_staging[((int64_t)threadIdx.x)];
  }
  mask[0] = __activemask();
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 16, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 8, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 4, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    ((volatile float*)red_result)[0] = red_buf0[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    T_softmax_maxelem_shared[0] = ((volatile float*)red_result)[0];
  }
  in_thread_T_softmax_expsum_shared[0] = 0.000000e+00f;
  __syncthreads();
  for (int64_t ax1_fused_0_1 = 0; ax1_fused_0_1 < ((n + (int64_t)1023) >> (int64_t)10); ++ax1_fused_0_1) {
    if (((ax1_fused_0_1 * (int64_t)1024) + ((int64_t)threadIdx.x)) < n) {
      in_thread_T_softmax_expsum_shared[0] = (in_thread_T_softmax_expsum_shared[0] + __expf((A[(((ax1_fused_0_1 * (int64_t)1024) + (((int64_t)blockIdx.x) * n)) + ((int64_t)threadIdx.x))] - T_softmax_maxelem_shared[0])));
    }
  }
  float red_buf0_2[1];
  uint mask_2[1];
  float t0_2[1];
  float red_buf0_3[1];
  uint mask_3[1];
  float t0_3[1];
  __shared__ float red_buf_staging_1[32];
  red_buf0_3[0] = in_thread_T_softmax_expsum_shared[0];
  mask_3[0] = __activemask();
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 16, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 8, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 4, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 2, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 1, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  if ((((int64_t)threadIdx.x) % (int64_t)32) == (int64_t)0) {
    red_buf_staging_1[(((int64_t)threadIdx.x) >> (int64_t)5)] = red_buf0_3[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) < (int64_t)32) {
    red_buf0_2[0] = red_buf_staging_1[((int64_t)threadIdx.x)];
  }
  mask_2[0] = __activemask();
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 16, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 8, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 4, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 2, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 1, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    ((volatile float*)red_result_1)[0] = red_buf0_2[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    T_softmax_expsum_shared[0] = ((volatile float*)red_result_1)[0];
  }
  __syncthreads();
  for (int64_t ax1_0 = 0; ax1_0 < ((n + (int64_t)1023) >> (int64_t)10); ++ax1_0) {
    if (((ax1_0 * (int64_t)1024) + ((int64_t)threadIdx.x)) < n) {
      compute_intermediate[(((ax1_0 * (int64_t)1024) + (((int64_t)blockIdx.x) * n)) + ((int64_t)threadIdx.x))] = ((half)(__expf((A[(((ax1_0 * (int64_t)1024) + (((int64_t)blockIdx.x) * n)) + ((int64_t)threadIdx.x))] - T_softmax_maxelem_shared[0])) / T_softmax_expsum_shared[0]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(128) fused_softmax_cast_n_1025_to_2048__kernel(float* __restrict__ A, half* __restrict__ compute_intermediate, int64_t n) {
  extern __shared__ float A_shared_dyn[];
  float in_thread_T_softmax_maxelem_shared[1];
  __shared__ float red_result[1];
  __shared__ float T_softmax_maxelem_shared[1];
  float in_thread_T_softmax_expsum_shared[1];
  __shared__ float red_result_1[1];
  __shared__ float T_softmax_expsum_shared[1];
  for (int64_t ax2_0 = 0; ax2_0 < ((n + (int64_t)511) >> (int64_t)9); ++ax2_0) {
    for (int64_t ax2_2_s = 0; ax2_2_s < (int64_t)4; ++ax2_2_s) {
      if (((((ax2_0 * (int64_t)512) + (((int64_t)threadIdx.x) * (int64_t)4)) + ax2_2_s) < n) && ((((ax2_0 * (int64_t)128) + ((int64_t)threadIdx.x)) - (((n + (int64_t)127) >> (int64_t)7) * (int64_t)32)) < (int64_t)0)) {
        A_shared_dyn[(((ax2_0 * (int64_t)512) + (((int64_t)threadIdx.x) * (int64_t)4)) + ax2_2_s)] = A[((((ax2_0 * (int64_t)512) + (((int64_t)threadIdx.x) * (int64_t)4)) + (((int64_t)blockIdx.x) * n)) + ax2_2_s)];
      }
    }
  }
  in_thread_T_softmax_maxelem_shared[0] = -3.402823e+38f;
  __syncthreads();
  for (int64_t ax1_fused_0 = 0; ax1_fused_0 < ((n + (int64_t)127) >> (int64_t)7); ++ax1_fused_0) {
    if (((ax1_fused_0 * (int64_t)128) + ((int64_t)threadIdx.x)) < n) {
      in_thread_T_softmax_maxelem_shared[0] = max(in_thread_T_softmax_maxelem_shared[0], A_shared_dyn[((ax1_fused_0 * (int64_t)128) + ((int64_t)threadIdx.x))]);
    }
  }
  float red_buf0[1];
  uint mask[1];
  float t0[1];
  float red_buf0_1[1];
  uint mask_1[1];
  float t0_1[1];
  __shared__ float red_buf_staging[4];
  red_buf0_1[0] = in_thread_T_softmax_maxelem_shared[0];
  mask_1[0] = __activemask();
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 16, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 8, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 4, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 2, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 1, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  if ((((int64_t)threadIdx.x) % (int64_t)32) == (int64_t)0) {
    red_buf_staging[(((int64_t)threadIdx.x) >> (int64_t)5)] = red_buf0_1[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) < (int64_t)4) {
    red_buf0[0] = red_buf_staging[((int64_t)threadIdx.x)];
  }
  mask[0] = (__activemask() & (uint)15);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    ((volatile float*)red_result)[0] = red_buf0[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    T_softmax_maxelem_shared[0] = ((volatile float*)red_result)[0];
  }
  in_thread_T_softmax_expsum_shared[0] = 0.000000e+00f;
  __syncthreads();
  for (int64_t ax1_fused_0_1 = 0; ax1_fused_0_1 < ((n + (int64_t)127) >> (int64_t)7); ++ax1_fused_0_1) {
    if (((ax1_fused_0_1 * (int64_t)128) + ((int64_t)threadIdx.x)) < n) {
      in_thread_T_softmax_expsum_shared[0] = (in_thread_T_softmax_expsum_shared[0] + __expf((A_shared_dyn[((ax1_fused_0_1 * (int64_t)128) + ((int64_t)threadIdx.x))] - T_softmax_maxelem_shared[0])));
    }
  }
  float red_buf0_2[1];
  uint mask_2[1];
  float t0_2[1];
  float red_buf0_3[1];
  uint mask_3[1];
  float t0_3[1];
  __shared__ float red_buf_staging_1[4];
  red_buf0_3[0] = in_thread_T_softmax_expsum_shared[0];
  mask_3[0] = __activemask();
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 16, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 8, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 4, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 2, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 1, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  if ((((int64_t)threadIdx.x) % (int64_t)32) == (int64_t)0) {
    red_buf_staging_1[(((int64_t)threadIdx.x) >> (int64_t)5)] = red_buf0_3[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) < (int64_t)4) {
    red_buf0_2[0] = red_buf_staging_1[((int64_t)threadIdx.x)];
  }
  mask_2[0] = (__activemask() & (uint)15);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 2, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 1, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    ((volatile float*)red_result_1)[0] = red_buf0_2[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    T_softmax_expsum_shared[0] = ((volatile float*)red_result_1)[0];
  }
  __syncthreads();
  for (int64_t ax1_0 = 0; ax1_0 < ((n + (int64_t)127) >> (int64_t)7); ++ax1_0) {
    if (((ax1_0 * (int64_t)128) + ((int64_t)threadIdx.x)) < n) {
      compute_intermediate[(((ax1_0 * (int64_t)128) + (((int64_t)blockIdx.x) * n)) + ((int64_t)threadIdx.x))] = ((half)(__expf((A_shared_dyn[((ax1_0 * (int64_t)128) + ((int64_t)threadIdx.x))] - T_softmax_maxelem_shared[0])) / T_softmax_expsum_shared[0]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(160) fused_softmax_cast_n_2049_to_2560__kernel(float* __restrict__ A, half* __restrict__ compute_intermediate, int64_t n) {
  extern __shared__ float A_shared_dyn[];
  float in_thread_T_softmax_maxelem_shared[1];
  __shared__ float red_result[1];
  __shared__ float T_softmax_maxelem_shared[1];
  float in_thread_T_softmax_expsum_shared[1];
  __shared__ float red_result_1[1];
  __shared__ float T_softmax_expsum_shared[1];
  for (int64_t ax2_0 = 0; ax2_0 < (((n + (int64_t)639) / (int64_t)640) + (((n + (int64_t)639) % (int64_t)640) >> (int64_t)63)); ++ax2_0) {
    for (int64_t ax2_2_s = 0; ax2_2_s < (int64_t)4; ++ax2_2_s) {
      if (((((ax2_0 * (int64_t)640) + (((int64_t)threadIdx.x) * (int64_t)4)) + ax2_2_s) < n) && ((((ax2_0 * (int64_t)160) + ((int64_t)threadIdx.x)) - ((((n + (int64_t)159) / (int64_t)160) + (((n + (int64_t)159) % (int64_t)160) >> (int64_t)63)) * (int64_t)40)) < (int64_t)0)) {
        A_shared_dyn[(((ax2_0 * (int64_t)640) + (((int64_t)threadIdx.x) * (int64_t)4)) + ax2_2_s)] = A[((((ax2_0 * (int64_t)640) + (((int64_t)threadIdx.x) * (int64_t)4)) + (((int64_t)blockIdx.x) * n)) + ax2_2_s)];
      }
    }
  }
  in_thread_T_softmax_maxelem_shared[0] = -3.402823e+38f;
  __syncthreads();
  for (int64_t ax1_fused_0 = 0; ax1_fused_0 < (((n + (int64_t)159) / (int64_t)160) + (((n + (int64_t)159) % (int64_t)160) >> (int64_t)63)); ++ax1_fused_0) {
    if (((ax1_fused_0 * (int64_t)160) + ((int64_t)threadIdx.x)) < n) {
      in_thread_T_softmax_maxelem_shared[0] = max(in_thread_T_softmax_maxelem_shared[0], A_shared_dyn[((ax1_fused_0 * (int64_t)160) + ((int64_t)threadIdx.x))]);
    }
  }
  float red_buf0[1];
  uint mask[1];
  float t0[1];
  float red_buf0_1[1];
  uint mask_1[1];
  float t0_1[1];
  __shared__ float red_buf_staging[5];
  red_buf0_1[0] = in_thread_T_softmax_maxelem_shared[0];
  mask_1[0] = __activemask();
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 16, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 8, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 4, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 2, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 1, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  if ((((int64_t)threadIdx.x) % (int64_t)32) == (int64_t)0) {
    red_buf_staging[(((int64_t)threadIdx.x) >> (int64_t)5)] = red_buf0_1[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) < (int64_t)5) {
    red_buf0[0] = red_buf_staging[((int64_t)threadIdx.x)];
  }
  mask[0] = (__activemask() & (uint)31);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 4, 32);
  if (((int64_t)threadIdx.x) < (int64_t)1) {
    red_buf0[0] = max(red_buf0[0], t0[0]);
  }
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    ((volatile float*)red_result)[0] = red_buf0[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    T_softmax_maxelem_shared[0] = ((volatile float*)red_result)[0];
  }
  in_thread_T_softmax_expsum_shared[0] = 0.000000e+00f;
  __syncthreads();
  for (int64_t ax1_fused_0_1 = 0; ax1_fused_0_1 < (((n + (int64_t)159) / (int64_t)160) + (((n + (int64_t)159) % (int64_t)160) >> (int64_t)63)); ++ax1_fused_0_1) {
    if (((ax1_fused_0_1 * (int64_t)160) + ((int64_t)threadIdx.x)) < n) {
      in_thread_T_softmax_expsum_shared[0] = (in_thread_T_softmax_expsum_shared[0] + __expf((A_shared_dyn[((ax1_fused_0_1 * (int64_t)160) + ((int64_t)threadIdx.x))] - T_softmax_maxelem_shared[0])));
    }
  }
  float red_buf0_2[1];
  uint mask_2[1];
  float t0_2[1];
  float red_buf0_3[1];
  uint mask_3[1];
  float t0_3[1];
  __shared__ float red_buf_staging_1[5];
  red_buf0_3[0] = in_thread_T_softmax_expsum_shared[0];
  mask_3[0] = __activemask();
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 16, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 8, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 4, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 2, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 1, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  if ((((int64_t)threadIdx.x) % (int64_t)32) == (int64_t)0) {
    red_buf_staging_1[(((int64_t)threadIdx.x) >> (int64_t)5)] = red_buf0_3[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) < (int64_t)5) {
    red_buf0_2[0] = red_buf_staging_1[((int64_t)threadIdx.x)];
  }
  mask_2[0] = (__activemask() & (uint)31);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 4, 32);
  if (((int64_t)threadIdx.x) < (int64_t)1) {
    red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  }
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 2, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 1, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    ((volatile float*)red_result_1)[0] = red_buf0_2[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    T_softmax_expsum_shared[0] = ((volatile float*)red_result_1)[0];
  }
  __syncthreads();
  for (int64_t ax1_0 = 0; ax1_0 < (((n + (int64_t)159) / (int64_t)160) + (((n + (int64_t)159) % (int64_t)160) >> (int64_t)63)); ++ax1_0) {
    if (((ax1_0 * (int64_t)160) + ((int64_t)threadIdx.x)) < n) {
      compute_intermediate[(((ax1_0 * (int64_t)160) + (((int64_t)blockIdx.x) * n)) + ((int64_t)threadIdx.x))] = ((half)(__expf((A_shared_dyn[((ax1_0 * (int64_t)160) + ((int64_t)threadIdx.x))] - T_softmax_maxelem_shared[0])) / T_softmax_expsum_shared[0]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(192) fused_softmax_cast_n_2561_to_3072__kernel(float* __restrict__ A, half* __restrict__ compute_intermediate, int64_t n) {
  extern __shared__ float A_shared_dyn[];
  float in_thread_T_softmax_maxelem_shared[1];
  __shared__ float red_result[1];
  __shared__ float T_softmax_maxelem_shared[1];
  float in_thread_T_softmax_expsum_shared[1];
  __shared__ float red_result_1[1];
  __shared__ float T_softmax_expsum_shared[1];
  for (int64_t ax2_0 = 0; ax2_0 < (((n + (int64_t)767) / (int64_t)768) + (((n + (int64_t)767) % (int64_t)768) >> (int64_t)63)); ++ax2_0) {
    for (int64_t ax2_2_s = 0; ax2_2_s < (int64_t)4; ++ax2_2_s) {
      if (((((ax2_0 * (int64_t)768) + (((int64_t)threadIdx.x) * (int64_t)4)) + ax2_2_s) < n) && ((((ax2_0 * (int64_t)192) + ((int64_t)threadIdx.x)) - ((((n + (int64_t)191) / (int64_t)192) + (((n + (int64_t)191) % (int64_t)192) >> (int64_t)63)) * (int64_t)48)) < (int64_t)0)) {
        A_shared_dyn[(((ax2_0 * (int64_t)768) + (((int64_t)threadIdx.x) * (int64_t)4)) + ax2_2_s)] = A[((((ax2_0 * (int64_t)768) + (((int64_t)threadIdx.x) * (int64_t)4)) + (((int64_t)blockIdx.x) * n)) + ax2_2_s)];
      }
    }
  }
  in_thread_T_softmax_maxelem_shared[0] = -3.402823e+38f;
  __syncthreads();
  for (int64_t ax1_fused_0 = 0; ax1_fused_0 < (((n + (int64_t)191) / (int64_t)192) + (((n + (int64_t)191) % (int64_t)192) >> (int64_t)63)); ++ax1_fused_0) {
    if (((ax1_fused_0 * (int64_t)192) + ((int64_t)threadIdx.x)) < n) {
      in_thread_T_softmax_maxelem_shared[0] = max(in_thread_T_softmax_maxelem_shared[0], A_shared_dyn[((ax1_fused_0 * (int64_t)192) + ((int64_t)threadIdx.x))]);
    }
  }
  float red_buf0[1];
  uint mask[1];
  float t0[1];
  float red_buf0_1[1];
  uint mask_1[1];
  float t0_1[1];
  __shared__ float red_buf_staging[6];
  red_buf0_1[0] = in_thread_T_softmax_maxelem_shared[0];
  mask_1[0] = __activemask();
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 16, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 8, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 4, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 2, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 1, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  if ((((int64_t)threadIdx.x) % (int64_t)32) == (int64_t)0) {
    red_buf_staging[(((int64_t)threadIdx.x) >> (int64_t)5)] = red_buf0_1[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) < (int64_t)6) {
    red_buf0[0] = red_buf_staging[((int64_t)threadIdx.x)];
  }
  mask[0] = (__activemask() & (uint)63);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 4, 32);
  if (((int64_t)threadIdx.x) < (int64_t)2) {
    red_buf0[0] = max(red_buf0[0], t0[0]);
  }
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    ((volatile float*)red_result)[0] = red_buf0[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    T_softmax_maxelem_shared[0] = ((volatile float*)red_result)[0];
  }
  in_thread_T_softmax_expsum_shared[0] = 0.000000e+00f;
  __syncthreads();
  for (int64_t ax1_fused_0_1 = 0; ax1_fused_0_1 < (((n + (int64_t)191) / (int64_t)192) + (((n + (int64_t)191) % (int64_t)192) >> (int64_t)63)); ++ax1_fused_0_1) {
    if (((ax1_fused_0_1 * (int64_t)192) + ((int64_t)threadIdx.x)) < n) {
      in_thread_T_softmax_expsum_shared[0] = (in_thread_T_softmax_expsum_shared[0] + __expf((A_shared_dyn[((ax1_fused_0_1 * (int64_t)192) + ((int64_t)threadIdx.x))] - T_softmax_maxelem_shared[0])));
    }
  }
  float red_buf0_2[1];
  uint mask_2[1];
  float t0_2[1];
  float red_buf0_3[1];
  uint mask_3[1];
  float t0_3[1];
  __shared__ float red_buf_staging_1[6];
  red_buf0_3[0] = in_thread_T_softmax_expsum_shared[0];
  mask_3[0] = __activemask();
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 16, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 8, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 4, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 2, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 1, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  if ((((int64_t)threadIdx.x) % (int64_t)32) == (int64_t)0) {
    red_buf_staging_1[(((int64_t)threadIdx.x) >> (int64_t)5)] = red_buf0_3[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) < (int64_t)6) {
    red_buf0_2[0] = red_buf_staging_1[((int64_t)threadIdx.x)];
  }
  mask_2[0] = (__activemask() & (uint)63);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 4, 32);
  if (((int64_t)threadIdx.x) < (int64_t)2) {
    red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  }
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 2, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 1, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    ((volatile float*)red_result_1)[0] = red_buf0_2[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    T_softmax_expsum_shared[0] = ((volatile float*)red_result_1)[0];
  }
  __syncthreads();
  for (int64_t ax1_0 = 0; ax1_0 < (((n + (int64_t)191) / (int64_t)192) + (((n + (int64_t)191) % (int64_t)192) >> (int64_t)63)); ++ax1_0) {
    if (((ax1_0 * (int64_t)192) + ((int64_t)threadIdx.x)) < n) {
      compute_intermediate[(((ax1_0 * (int64_t)192) + (((int64_t)blockIdx.x) * n)) + ((int64_t)threadIdx.x))] = ((half)(__expf((A_shared_dyn[((ax1_0 * (int64_t)192) + ((int64_t)threadIdx.x))] - T_softmax_maxelem_shared[0])) / T_softmax_expsum_shared[0]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(192) fused_softmax_cast_n_3073_to_3584__kernel(float* __restrict__ A, half* __restrict__ compute_intermediate, int64_t n) {
  extern __shared__ float A_shared_dyn[];
  float in_thread_T_softmax_maxelem_shared[1];
  __shared__ float red_result[1];
  __shared__ float T_softmax_maxelem_shared[1];
  float in_thread_T_softmax_expsum_shared[1];
  __shared__ float red_result_1[1];
  __shared__ float T_softmax_expsum_shared[1];
  for (int64_t ax2_0 = 0; ax2_0 < (((n + (int64_t)767) / (int64_t)768) + (((n + (int64_t)767) % (int64_t)768) >> (int64_t)63)); ++ax2_0) {
    for (int64_t ax2_2_s = 0; ax2_2_s < (int64_t)4; ++ax2_2_s) {
      if (((((ax2_0 * (int64_t)768) + (((int64_t)threadIdx.x) * (int64_t)4)) + ax2_2_s) < n) && ((((ax2_0 * (int64_t)192) + ((int64_t)threadIdx.x)) - ((((n + (int64_t)191) / (int64_t)192) + (((n + (int64_t)191) % (int64_t)192) >> (int64_t)63)) * (int64_t)48)) < (int64_t)0)) {
        A_shared_dyn[(((ax2_0 * (int64_t)768) + (((int64_t)threadIdx.x) * (int64_t)4)) + ax2_2_s)] = A[((((ax2_0 * (int64_t)768) + (((int64_t)threadIdx.x) * (int64_t)4)) + (((int64_t)blockIdx.x) * n)) + ax2_2_s)];
      }
    }
  }
  in_thread_T_softmax_maxelem_shared[0] = -3.402823e+38f;
  __syncthreads();
  for (int64_t ax1_fused_0 = 0; ax1_fused_0 < (((n + (int64_t)191) / (int64_t)192) + (((n + (int64_t)191) % (int64_t)192) >> (int64_t)63)); ++ax1_fused_0) {
    if (((ax1_fused_0 * (int64_t)192) + ((int64_t)threadIdx.x)) < n) {
      in_thread_T_softmax_maxelem_shared[0] = max(in_thread_T_softmax_maxelem_shared[0], A_shared_dyn[((ax1_fused_0 * (int64_t)192) + ((int64_t)threadIdx.x))]);
    }
  }
  float red_buf0[1];
  uint mask[1];
  float t0[1];
  float red_buf0_1[1];
  uint mask_1[1];
  float t0_1[1];
  __shared__ float red_buf_staging[6];
  red_buf0_1[0] = in_thread_T_softmax_maxelem_shared[0];
  mask_1[0] = __activemask();
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 16, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 8, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 4, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 2, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 1, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  if ((((int64_t)threadIdx.x) % (int64_t)32) == (int64_t)0) {
    red_buf_staging[(((int64_t)threadIdx.x) >> (int64_t)5)] = red_buf0_1[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) < (int64_t)6) {
    red_buf0[0] = red_buf_staging[((int64_t)threadIdx.x)];
  }
  mask[0] = (__activemask() & (uint)63);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 4, 32);
  if (((int64_t)threadIdx.x) < (int64_t)2) {
    red_buf0[0] = max(red_buf0[0], t0[0]);
  }
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    ((volatile float*)red_result)[0] = red_buf0[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    T_softmax_maxelem_shared[0] = ((volatile float*)red_result)[0];
  }
  in_thread_T_softmax_expsum_shared[0] = 0.000000e+00f;
  __syncthreads();
  for (int64_t ax1_fused_0_1 = 0; ax1_fused_0_1 < (((n + (int64_t)191) / (int64_t)192) + (((n + (int64_t)191) % (int64_t)192) >> (int64_t)63)); ++ax1_fused_0_1) {
    if (((ax1_fused_0_1 * (int64_t)192) + ((int64_t)threadIdx.x)) < n) {
      in_thread_T_softmax_expsum_shared[0] = (in_thread_T_softmax_expsum_shared[0] + __expf((A_shared_dyn[((ax1_fused_0_1 * (int64_t)192) + ((int64_t)threadIdx.x))] - T_softmax_maxelem_shared[0])));
    }
  }
  float red_buf0_2[1];
  uint mask_2[1];
  float t0_2[1];
  float red_buf0_3[1];
  uint mask_3[1];
  float t0_3[1];
  __shared__ float red_buf_staging_1[6];
  red_buf0_3[0] = in_thread_T_softmax_expsum_shared[0];
  mask_3[0] = __activemask();
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 16, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 8, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 4, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 2, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 1, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  if ((((int64_t)threadIdx.x) % (int64_t)32) == (int64_t)0) {
    red_buf_staging_1[(((int64_t)threadIdx.x) >> (int64_t)5)] = red_buf0_3[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) < (int64_t)6) {
    red_buf0_2[0] = red_buf_staging_1[((int64_t)threadIdx.x)];
  }
  mask_2[0] = (__activemask() & (uint)63);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 4, 32);
  if (((int64_t)threadIdx.x) < (int64_t)2) {
    red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  }
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 2, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 1, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    ((volatile float*)red_result_1)[0] = red_buf0_2[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    T_softmax_expsum_shared[0] = ((volatile float*)red_result_1)[0];
  }
  __syncthreads();
  for (int64_t ax1_0 = 0; ax1_0 < (((n + (int64_t)191) / (int64_t)192) + (((n + (int64_t)191) % (int64_t)192) >> (int64_t)63)); ++ax1_0) {
    if (((ax1_0 * (int64_t)192) + ((int64_t)threadIdx.x)) < n) {
      compute_intermediate[(((ax1_0 * (int64_t)192) + (((int64_t)blockIdx.x) * n)) + ((int64_t)threadIdx.x))] = ((half)(__expf((A_shared_dyn[((ax1_0 * (int64_t)192) + ((int64_t)threadIdx.x))] - T_softmax_maxelem_shared[0])) / T_softmax_expsum_shared[0]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) fused_softmax_cast_n_3585_to_4096__kernel(float* __restrict__ A, half* __restrict__ compute_intermediate, int64_t n) {
  extern __shared__ float A_shared_dyn[];
  float in_thread_T_softmax_maxelem_shared[1];
  __shared__ float red_result[1];
  __shared__ float T_softmax_maxelem_shared[1];
  float in_thread_T_softmax_expsum_shared[1];
  __shared__ float red_result_1[1];
  __shared__ float T_softmax_expsum_shared[1];
  for (int64_t ax2_0 = 0; ax2_0 < ((n + (int64_t)1023) >> (int64_t)10); ++ax2_0) {
    for (int64_t ax2_2_s = 0; ax2_2_s < (int64_t)4; ++ax2_2_s) {
      if (((((ax2_0 * (int64_t)1024) + (((int64_t)threadIdx.x) * (int64_t)4)) + ax2_2_s) < n) && ((((ax2_0 * (int64_t)256) + ((int64_t)threadIdx.x)) - (((n + (int64_t)255) >> (int64_t)8) * (int64_t)64)) < (int64_t)0)) {
        A_shared_dyn[(((ax2_0 * (int64_t)1024) + (((int64_t)threadIdx.x) * (int64_t)4)) + ax2_2_s)] = A[((((ax2_0 * (int64_t)1024) + (((int64_t)threadIdx.x) * (int64_t)4)) + (((int64_t)blockIdx.x) * n)) + ax2_2_s)];
      }
    }
  }
  in_thread_T_softmax_maxelem_shared[0] = -3.402823e+38f;
  __syncthreads();
  for (int64_t ax1_fused_0 = 0; ax1_fused_0 < ((n + (int64_t)255) >> (int64_t)8); ++ax1_fused_0) {
    if (((ax1_fused_0 * (int64_t)256) + ((int64_t)threadIdx.x)) < n) {
      in_thread_T_softmax_maxelem_shared[0] = max(in_thread_T_softmax_maxelem_shared[0], A_shared_dyn[((ax1_fused_0 * (int64_t)256) + ((int64_t)threadIdx.x))]);
    }
  }
  float red_buf0[1];
  uint mask[1];
  float t0[1];
  float red_buf0_1[1];
  uint mask_1[1];
  float t0_1[1];
  __shared__ float red_buf_staging[8];
  red_buf0_1[0] = in_thread_T_softmax_maxelem_shared[0];
  mask_1[0] = __activemask();
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 16, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 8, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 4, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 2, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 1, 32);
  red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
  if ((((int64_t)threadIdx.x) % (int64_t)32) == (int64_t)0) {
    red_buf_staging[(((int64_t)threadIdx.x) >> (int64_t)5)] = red_buf0_1[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) < (int64_t)8) {
    red_buf0[0] = red_buf_staging[((int64_t)threadIdx.x)];
  }
  mask[0] = (__activemask() & (uint)255);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 4, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
  red_buf0[0] = max(red_buf0[0], t0[0]);
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    ((volatile float*)red_result)[0] = red_buf0[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    T_softmax_maxelem_shared[0] = ((volatile float*)red_result)[0];
  }
  in_thread_T_softmax_expsum_shared[0] = 0.000000e+00f;
  __syncthreads();
  for (int64_t ax1_fused_0_1 = 0; ax1_fused_0_1 < ((n + (int64_t)255) >> (int64_t)8); ++ax1_fused_0_1) {
    if (((ax1_fused_0_1 * (int64_t)256) + ((int64_t)threadIdx.x)) < n) {
      in_thread_T_softmax_expsum_shared[0] = (in_thread_T_softmax_expsum_shared[0] + __expf((A_shared_dyn[((ax1_fused_0_1 * (int64_t)256) + ((int64_t)threadIdx.x))] - T_softmax_maxelem_shared[0])));
    }
  }
  float red_buf0_2[1];
  uint mask_2[1];
  float t0_2[1];
  float red_buf0_3[1];
  uint mask_3[1];
  float t0_3[1];
  __shared__ float red_buf_staging_1[8];
  red_buf0_3[0] = in_thread_T_softmax_expsum_shared[0];
  mask_3[0] = __activemask();
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 16, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 8, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 4, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 2, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  t0_3[0] = __shfl_down_sync(mask_3[0], red_buf0_3[0], 1, 32);
  red_buf0_3[0] = (red_buf0_3[0] + t0_3[0]);
  if ((((int64_t)threadIdx.x) % (int64_t)32) == (int64_t)0) {
    red_buf_staging_1[(((int64_t)threadIdx.x) >> (int64_t)5)] = red_buf0_3[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) < (int64_t)8) {
    red_buf0_2[0] = red_buf_staging_1[((int64_t)threadIdx.x)];
  }
  mask_2[0] = (__activemask() & (uint)255);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 4, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 2, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 1, 32);
  red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    ((volatile float*)red_result_1)[0] = red_buf0_2[0];
  }
  __syncthreads();
  if (((int64_t)threadIdx.x) == (int64_t)0) {
    T_softmax_expsum_shared[0] = ((volatile float*)red_result_1)[0];
  }
  __syncthreads();
  for (int64_t ax1_0 = 0; ax1_0 < ((n + (int64_t)255) >> (int64_t)8); ++ax1_0) {
    if (((ax1_0 * (int64_t)256) + ((int64_t)threadIdx.x)) < n) {
      compute_intermediate[(((ax1_0 * (int64_t)256) + (((int64_t)blockIdx.x) * n)) + ((int64_t)threadIdx.x))] = ((half)(__expf((A_shared_dyn[((ax1_0 * (int64_t)256) + ((int64_t)threadIdx.x))] - T_softmax_maxelem_shared[0])) / T_softmax_expsum_shared[0]));
    }
  }
}

