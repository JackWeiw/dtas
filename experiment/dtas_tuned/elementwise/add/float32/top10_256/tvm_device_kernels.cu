#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <math.h>
#include <mma.h>
//for int8
#include <sm_61_intrinsics.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) ||      (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) add_n_1_to_256__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_1_to_256_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_257_to_512__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_257_to_512_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_513_to_768__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_513_to_768_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_769_to_1024__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_769_to_1024_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_1025_to_1280__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_1025_to_1280_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_1281_to_1536__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_1281_to_1536_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_1537_to_1792__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_1537_to_1792_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_1793_to_2048__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_1793_to_2048_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_2049_to_2304__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_2049_to_2304_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_2305_to_2560__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_2305_to_2560_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_2561_to_2816__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_2561_to_2816_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_2817_to_3072__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_2817_to_3072_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_3073_to_3328__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_3073_to_3328_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_3329_to_3584__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_3329_to_3584_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_3585_to_3840__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_3585_to_3840_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_3841_to_4096__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_3841_to_4096_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_1_to_256__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_1_to_256_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < ((n + (int64_t)255) >> (int64_t)8); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * ((n + (int64_t)255) >> (int64_t)8)) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)640)) < (int64_t)0) {
      float4 __1;
        float4 v_ = *(float4*)(A + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * ((n + (int64_t)255) >> (int64_t)8)) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        float4 v__1 = *(float4*)(B + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * ((n + (int64_t)255) >> (int64_t)8)) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        __1.x = (v_.x+v__1.x);
        __1.y = (v_.y+v__1.y);
        __1.z = (v_.z+v__1.z);
        __1.w = (v_.w+v__1.w);
      *(float4*)(T_add_n_1_to_256_ + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * ((n + (int64_t)255) >> (int64_t)8)) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_257_to_512__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_257_to_512_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < ((n + (int64_t)511) >> (int64_t)9); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * ((n + (int64_t)511) >> (int64_t)9)) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)640)) < (int64_t)0) {
      float4 __1;
        float4 v_ = *(float4*)(A + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * ((n + (int64_t)511) >> (int64_t)9)) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        float4 v__1 = *(float4*)(B + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * ((n + (int64_t)511) >> (int64_t)9)) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        __1.x = (v_.x+v__1.x);
        __1.y = (v_.y+v__1.y);
        __1.z = (v_.z+v__1.z);
        __1.w = (v_.w+v__1.w);
      *(float4*)(T_add_n_257_to_512_ + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * ((n + (int64_t)511) >> (int64_t)9)) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_513_to_768__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_513_to_768_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)767) / (int64_t)768) + (((n + (int64_t)767) % (int64_t)768) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)767) / (int64_t)768) + (((n + (int64_t)767) % (int64_t)768) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)640)) < (int64_t)0) {
      float4 __1;
        float4 v_ = *(float4*)(A + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)767) / (int64_t)768) + (((n + (int64_t)767) % (int64_t)768) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        float4 v__1 = *(float4*)(B + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)767) / (int64_t)768) + (((n + (int64_t)767) % (int64_t)768) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        __1.x = (v_.x+v__1.x);
        __1.y = (v_.y+v__1.y);
        __1.z = (v_.z+v__1.z);
        __1.w = (v_.w+v__1.w);
      *(float4*)(T_add_n_513_to_768_ + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)767) / (int64_t)768) + (((n + (int64_t)767) % (int64_t)768) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_769_to_1024__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_769_to_1024_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < ((n + (int64_t)1023) >> (int64_t)10); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * ((n + (int64_t)1023) >> (int64_t)10)) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)640)) < (int64_t)0) {
      float4 __1;
        float4 v_ = *(float4*)(A + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * ((n + (int64_t)1023) >> (int64_t)10)) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        float4 v__1 = *(float4*)(B + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * ((n + (int64_t)1023) >> (int64_t)10)) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        __1.x = (v_.x+v__1.x);
        __1.y = (v_.y+v__1.y);
        __1.z = (v_.z+v__1.z);
        __1.w = (v_.w+v__1.w);
      *(float4*)(T_add_n_769_to_1024_ + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * ((n + (int64_t)1023) >> (int64_t)10)) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_1025_to_1280__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_1025_to_1280_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)1279) / (int64_t)1280) + (((n + (int64_t)1279) % (int64_t)1280) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)1279) / (int64_t)1280) + (((n + (int64_t)1279) % (int64_t)1280) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)640)) < (int64_t)0) {
      float4 __1;
        float4 v_ = *(float4*)(A + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1279) / (int64_t)1280) + (((n + (int64_t)1279) % (int64_t)1280) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        float4 v__1 = *(float4*)(B + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1279) / (int64_t)1280) + (((n + (int64_t)1279) % (int64_t)1280) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        __1.x = (v_.x+v__1.x);
        __1.y = (v_.y+v__1.y);
        __1.z = (v_.z+v__1.z);
        __1.w = (v_.w+v__1.w);
      *(float4*)(T_add_n_1025_to_1280_ + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1279) / (int64_t)1280) + (((n + (int64_t)1279) % (int64_t)1280) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_1281_to_1536__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_1281_to_1536_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)1535) / (int64_t)1536) + (((n + (int64_t)1535) % (int64_t)1536) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)1535) / (int64_t)1536) + (((n + (int64_t)1535) % (int64_t)1536) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)640)) < (int64_t)0) {
      float4 __1;
        float4 v_ = *(float4*)(A + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1535) / (int64_t)1536) + (((n + (int64_t)1535) % (int64_t)1536) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        float4 v__1 = *(float4*)(B + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1535) / (int64_t)1536) + (((n + (int64_t)1535) % (int64_t)1536) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        __1.x = (v_.x+v__1.x);
        __1.y = (v_.y+v__1.y);
        __1.z = (v_.z+v__1.z);
        __1.w = (v_.w+v__1.w);
      *(float4*)(T_add_n_1281_to_1536_ + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1535) / (int64_t)1536) + (((n + (int64_t)1535) % (int64_t)1536) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_1537_to_1792__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_1537_to_1792_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)1791) / (int64_t)1792) + (((n + (int64_t)1791) % (int64_t)1792) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)1791) / (int64_t)1792) + (((n + (int64_t)1791) % (int64_t)1792) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)640)) < (int64_t)0) {
      float4 __1;
        float4 v_ = *(float4*)(A + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1791) / (int64_t)1792) + (((n + (int64_t)1791) % (int64_t)1792) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        float4 v__1 = *(float4*)(B + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1791) / (int64_t)1792) + (((n + (int64_t)1791) % (int64_t)1792) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        __1.x = (v_.x+v__1.x);
        __1.y = (v_.y+v__1.y);
        __1.z = (v_.z+v__1.z);
        __1.w = (v_.w+v__1.w);
      *(float4*)(T_add_n_1537_to_1792_ + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1791) / (int64_t)1792) + (((n + (int64_t)1791) % (int64_t)1792) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_1793_to_2048__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_1793_to_2048_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < ((n + (int64_t)2047) >> (int64_t)11); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * ((n + (int64_t)2047) >> (int64_t)11)) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)640)) < (int64_t)0) {
      float4 __1;
        float4 v_ = *(float4*)(A + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * ((n + (int64_t)2047) >> (int64_t)11)) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        float4 v__1 = *(float4*)(B + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * ((n + (int64_t)2047) >> (int64_t)11)) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        __1.x = (v_.x+v__1.x);
        __1.y = (v_.y+v__1.y);
        __1.z = (v_.z+v__1.z);
        __1.w = (v_.w+v__1.w);
      *(float4*)(T_add_n_1793_to_2048_ + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * ((n + (int64_t)2047) >> (int64_t)11)) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_2049_to_2304__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_2049_to_2304_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)2303) / (int64_t)2304) + (((n + (int64_t)2303) % (int64_t)2304) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)2303) / (int64_t)2304) + (((n + (int64_t)2303) % (int64_t)2304) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)640)) < (int64_t)0) {
      float4 __1;
        float4 v_ = *(float4*)(A + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2303) / (int64_t)2304) + (((n + (int64_t)2303) % (int64_t)2304) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        float4 v__1 = *(float4*)(B + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2303) / (int64_t)2304) + (((n + (int64_t)2303) % (int64_t)2304) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        __1.x = (v_.x+v__1.x);
        __1.y = (v_.y+v__1.y);
        __1.z = (v_.z+v__1.z);
        __1.w = (v_.w+v__1.w);
      *(float4*)(T_add_n_2049_to_2304_ + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2303) / (int64_t)2304) + (((n + (int64_t)2303) % (int64_t)2304) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_2305_to_2560__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_2305_to_2560_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)2559) / (int64_t)2560) + (((n + (int64_t)2559) % (int64_t)2560) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)2559) / (int64_t)2560) + (((n + (int64_t)2559) % (int64_t)2560) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)640)) < (int64_t)0) {
      float4 __1;
        float4 v_ = *(float4*)(A + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2559) / (int64_t)2560) + (((n + (int64_t)2559) % (int64_t)2560) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        float4 v__1 = *(float4*)(B + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2559) / (int64_t)2560) + (((n + (int64_t)2559) % (int64_t)2560) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        __1.x = (v_.x+v__1.x);
        __1.y = (v_.y+v__1.y);
        __1.z = (v_.z+v__1.z);
        __1.w = (v_.w+v__1.w);
      *(float4*)(T_add_n_2305_to_2560_ + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2559) / (int64_t)2560) + (((n + (int64_t)2559) % (int64_t)2560) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_2561_to_2816__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_2561_to_2816_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)2815) / (int64_t)2816) + (((n + (int64_t)2815) % (int64_t)2816) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)2815) / (int64_t)2816) + (((n + (int64_t)2815) % (int64_t)2816) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)640)) < (int64_t)0) {
      float4 __1;
        float4 v_ = *(float4*)(A + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2815) / (int64_t)2816) + (((n + (int64_t)2815) % (int64_t)2816) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        float4 v__1 = *(float4*)(B + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2815) / (int64_t)2816) + (((n + (int64_t)2815) % (int64_t)2816) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        __1.x = (v_.x+v__1.x);
        __1.y = (v_.y+v__1.y);
        __1.z = (v_.z+v__1.z);
        __1.w = (v_.w+v__1.w);
      *(float4*)(T_add_n_2561_to_2816_ + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2815) / (int64_t)2816) + (((n + (int64_t)2815) % (int64_t)2816) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_2817_to_3072__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_2817_to_3072_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)3071) / (int64_t)3072) + (((n + (int64_t)3071) % (int64_t)3072) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)3071) / (int64_t)3072) + (((n + (int64_t)3071) % (int64_t)3072) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)640)) < (int64_t)0) {
      float4 __1;
        float4 v_ = *(float4*)(A + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3071) / (int64_t)3072) + (((n + (int64_t)3071) % (int64_t)3072) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        float4 v__1 = *(float4*)(B + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3071) / (int64_t)3072) + (((n + (int64_t)3071) % (int64_t)3072) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        __1.x = (v_.x+v__1.x);
        __1.y = (v_.y+v__1.y);
        __1.z = (v_.z+v__1.z);
        __1.w = (v_.w+v__1.w);
      *(float4*)(T_add_n_2817_to_3072_ + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3071) / (int64_t)3072) + (((n + (int64_t)3071) % (int64_t)3072) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_3073_to_3328__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_3073_to_3328_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)3327) / (int64_t)3328) + (((n + (int64_t)3327) % (int64_t)3328) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)3327) / (int64_t)3328) + (((n + (int64_t)3327) % (int64_t)3328) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)640)) < (int64_t)0) {
      float4 __1;
        float4 v_ = *(float4*)(A + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3327) / (int64_t)3328) + (((n + (int64_t)3327) % (int64_t)3328) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        float4 v__1 = *(float4*)(B + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3327) / (int64_t)3328) + (((n + (int64_t)3327) % (int64_t)3328) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        __1.x = (v_.x+v__1.x);
        __1.y = (v_.y+v__1.y);
        __1.z = (v_.z+v__1.z);
        __1.w = (v_.w+v__1.w);
      *(float4*)(T_add_n_3073_to_3328_ + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3327) / (int64_t)3328) + (((n + (int64_t)3327) % (int64_t)3328) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_3329_to_3584__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_3329_to_3584_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)3583) / (int64_t)3584) + (((n + (int64_t)3583) % (int64_t)3584) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)3583) / (int64_t)3584) + (((n + (int64_t)3583) % (int64_t)3584) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)640)) < (int64_t)0) {
      float4 __1;
        float4 v_ = *(float4*)(A + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3583) / (int64_t)3584) + (((n + (int64_t)3583) % (int64_t)3584) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        float4 v__1 = *(float4*)(B + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3583) / (int64_t)3584) + (((n + (int64_t)3583) % (int64_t)3584) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        __1.x = (v_.x+v__1.x);
        __1.y = (v_.y+v__1.y);
        __1.z = (v_.z+v__1.z);
        __1.w = (v_.w+v__1.w);
      *(float4*)(T_add_n_3329_to_3584_ + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3583) / (int64_t)3584) + (((n + (int64_t)3583) % (int64_t)3584) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_3585_to_3840__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_3585_to_3840_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)3839) / (int64_t)3840) + (((n + (int64_t)3839) % (int64_t)3840) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)3839) / (int64_t)3840) + (((n + (int64_t)3839) % (int64_t)3840) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)640)) < (int64_t)0) {
      float4 __1;
        float4 v_ = *(float4*)(A + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3839) / (int64_t)3840) + (((n + (int64_t)3839) % (int64_t)3840) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        float4 v__1 = *(float4*)(B + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3839) / (int64_t)3840) + (((n + (int64_t)3839) % (int64_t)3840) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        __1.x = (v_.x+v__1.x);
        __1.y = (v_.y+v__1.y);
        __1.z = (v_.z+v__1.z);
        __1.w = (v_.w+v__1.w);
      *(float4*)(T_add_n_3585_to_3840_ + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3839) / (int64_t)3840) + (((n + (int64_t)3839) % (int64_t)3840) >> (int64_t)63))) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_3841_to_4096__kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add_n_3841_to_4096_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < ((n + (int64_t)4095) >> (int64_t)12); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * ((n + (int64_t)4095) >> (int64_t)12)) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)640)) < (int64_t)0) {
      float4 __1;
        float4 v_ = *(float4*)(A + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * ((n + (int64_t)4095) >> (int64_t)12)) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        float4 v__1 = *(float4*)(B + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * ((n + (int64_t)4095) >> (int64_t)12)) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4)));
        __1.x = (v_.x+v__1.x);
        __1.y = (v_.y+v__1.y);
        __1.z = (v_.z+v__1.z);
        __1.w = (v_.w+v__1.w);
      *(float4*)(T_add_n_3841_to_4096_ + (((ax0_ax1_fused_1 * (int64_t)1024) + ((((int64_t)blockIdx.x) * ((n + (int64_t)4095) >> (int64_t)12)) * (int64_t)1024)) + (((int64_t)threadIdx.x) * (int64_t)4))) = __1;
    }
  }
}

