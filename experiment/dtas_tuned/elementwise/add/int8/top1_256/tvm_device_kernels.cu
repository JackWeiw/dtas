#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <math.h>
#include <mma.h>
//for int8
#include <sm_61_intrinsics.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) ||      (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) add_n_1_to_256__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_1_to_256_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_257_to_512__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_257_to_512_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_513_to_768__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_513_to_768_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_769_to_1024__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_769_to_1024_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_1025_to_1280__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_1025_to_1280_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_1281_to_1536__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_1281_to_1536_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_1537_to_1792__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_1537_to_1792_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_1793_to_2048__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_1793_to_2048_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_2049_to_2304__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_2049_to_2304_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_2305_to_2560__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_2305_to_2560_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_2561_to_2816__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_2561_to_2816_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_2817_to_3072__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_2817_to_3072_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_3073_to_3328__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_3073_to_3328_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_3329_to_3584__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_3329_to_3584_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_3585_to_3840__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_3585_to_3840_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_3841_to_4096__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_3841_to_4096_, int64_t n);
extern "C" __global__ void __launch_bounds__(256) add_n_1_to_256__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_1_to_256_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < ((n + (int64_t)255) >> (int64_t)8); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * ((n + (int64_t)255) >> (int64_t)8)) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)160)) < (int64_t)0) {
      int4 __1;
        int4 v_ = *(int4*)(A + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * ((n + (int64_t)255) >> (int64_t)8)) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        int4 v__1 = *(int4*)(B + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * ((n + (int64_t)255) >> (int64_t)8)) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        __1.x=((((char)(v_.x >> 0))+((char)(v__1.x >> 0))) << 0);
        __1.x=__1.x & ~(0x000000ff << 8) |((((char)(v_.x >> 8))+((char)(v__1.x >> 8))) << 8);
        __1.x=__1.x & ~(0x000000ff << 16) |((((char)(v_.x >> 16))+((char)(v__1.x >> 16))) << 16);
        __1.x=__1.x & ~(0x000000ff << 24) |((((char)(v_.x >> 24))+((char)(v__1.x >> 24))) << 24);
        __1.y=__1.y & ~(0x000000ff << 0) |((((char)(v_.y >> 0))+((char)(v__1.y >> 0))) << 0);
        __1.y=__1.y & ~(0x000000ff << 8) |((((char)(v_.y >> 8))+((char)(v__1.y >> 8))) << 8);
        __1.y=__1.y & ~(0x000000ff << 16) |((((char)(v_.y >> 16))+((char)(v__1.y >> 16))) << 16);
        __1.y=__1.y & ~(0x000000ff << 24) |((((char)(v_.y >> 24))+((char)(v__1.y >> 24))) << 24);
        __1.z=__1.z & ~(0x000000ff << 0) |((((char)(v_.z >> 0))+((char)(v__1.z >> 0))) << 0);
        __1.z=__1.z & ~(0x000000ff << 8) |((((char)(v_.z >> 8))+((char)(v__1.z >> 8))) << 8);
        __1.z=__1.z & ~(0x000000ff << 16) |((((char)(v_.z >> 16))+((char)(v__1.z >> 16))) << 16);
        __1.z=__1.z & ~(0x000000ff << 24) |((((char)(v_.z >> 24))+((char)(v__1.z >> 24))) << 24);
        __1.w=__1.w & ~(0x000000ff << 0) |((((char)(v_.w >> 0))+((char)(v__1.w >> 0))) << 0);
        __1.w=__1.w & ~(0x000000ff << 8) |((((char)(v_.w >> 8))+((char)(v__1.w >> 8))) << 8);
        __1.w=__1.w & ~(0x000000ff << 16) |((((char)(v_.w >> 16))+((char)(v__1.w >> 16))) << 16);
        __1.w=__1.w & ~(0x000000ff << 24) |((((char)(v_.w >> 24))+((char)(v__1.w >> 24))) << 24);
      *(int4*)(T_add_n_1_to_256_ + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * ((n + (int64_t)255) >> (int64_t)8)) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_257_to_512__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_257_to_512_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < ((n + (int64_t)511) >> (int64_t)9); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * ((n + (int64_t)511) >> (int64_t)9)) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)160)) < (int64_t)0) {
      int4 __1;
        int4 v_ = *(int4*)(A + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * ((n + (int64_t)511) >> (int64_t)9)) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        int4 v__1 = *(int4*)(B + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * ((n + (int64_t)511) >> (int64_t)9)) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        __1.x=((((char)(v_.x >> 0))+((char)(v__1.x >> 0))) << 0);
        __1.x=__1.x & ~(0x000000ff << 8) |((((char)(v_.x >> 8))+((char)(v__1.x >> 8))) << 8);
        __1.x=__1.x & ~(0x000000ff << 16) |((((char)(v_.x >> 16))+((char)(v__1.x >> 16))) << 16);
        __1.x=__1.x & ~(0x000000ff << 24) |((((char)(v_.x >> 24))+((char)(v__1.x >> 24))) << 24);
        __1.y=__1.y & ~(0x000000ff << 0) |((((char)(v_.y >> 0))+((char)(v__1.y >> 0))) << 0);
        __1.y=__1.y & ~(0x000000ff << 8) |((((char)(v_.y >> 8))+((char)(v__1.y >> 8))) << 8);
        __1.y=__1.y & ~(0x000000ff << 16) |((((char)(v_.y >> 16))+((char)(v__1.y >> 16))) << 16);
        __1.y=__1.y & ~(0x000000ff << 24) |((((char)(v_.y >> 24))+((char)(v__1.y >> 24))) << 24);
        __1.z=__1.z & ~(0x000000ff << 0) |((((char)(v_.z >> 0))+((char)(v__1.z >> 0))) << 0);
        __1.z=__1.z & ~(0x000000ff << 8) |((((char)(v_.z >> 8))+((char)(v__1.z >> 8))) << 8);
        __1.z=__1.z & ~(0x000000ff << 16) |((((char)(v_.z >> 16))+((char)(v__1.z >> 16))) << 16);
        __1.z=__1.z & ~(0x000000ff << 24) |((((char)(v_.z >> 24))+((char)(v__1.z >> 24))) << 24);
        __1.w=__1.w & ~(0x000000ff << 0) |((((char)(v_.w >> 0))+((char)(v__1.w >> 0))) << 0);
        __1.w=__1.w & ~(0x000000ff << 8) |((((char)(v_.w >> 8))+((char)(v__1.w >> 8))) << 8);
        __1.w=__1.w & ~(0x000000ff << 16) |((((char)(v_.w >> 16))+((char)(v__1.w >> 16))) << 16);
        __1.w=__1.w & ~(0x000000ff << 24) |((((char)(v_.w >> 24))+((char)(v__1.w >> 24))) << 24);
      *(int4*)(T_add_n_257_to_512_ + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * ((n + (int64_t)511) >> (int64_t)9)) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_513_to_768__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_513_to_768_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)767) / (int64_t)768) + (((n + (int64_t)767) % (int64_t)768) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)767) / (int64_t)768) + (((n + (int64_t)767) % (int64_t)768) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)160)) < (int64_t)0) {
      int4 __1;
        int4 v_ = *(int4*)(A + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)767) / (int64_t)768) + (((n + (int64_t)767) % (int64_t)768) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        int4 v__1 = *(int4*)(B + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)767) / (int64_t)768) + (((n + (int64_t)767) % (int64_t)768) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        __1.x=((((char)(v_.x >> 0))+((char)(v__1.x >> 0))) << 0);
        __1.x=__1.x & ~(0x000000ff << 8) |((((char)(v_.x >> 8))+((char)(v__1.x >> 8))) << 8);
        __1.x=__1.x & ~(0x000000ff << 16) |((((char)(v_.x >> 16))+((char)(v__1.x >> 16))) << 16);
        __1.x=__1.x & ~(0x000000ff << 24) |((((char)(v_.x >> 24))+((char)(v__1.x >> 24))) << 24);
        __1.y=__1.y & ~(0x000000ff << 0) |((((char)(v_.y >> 0))+((char)(v__1.y >> 0))) << 0);
        __1.y=__1.y & ~(0x000000ff << 8) |((((char)(v_.y >> 8))+((char)(v__1.y >> 8))) << 8);
        __1.y=__1.y & ~(0x000000ff << 16) |((((char)(v_.y >> 16))+((char)(v__1.y >> 16))) << 16);
        __1.y=__1.y & ~(0x000000ff << 24) |((((char)(v_.y >> 24))+((char)(v__1.y >> 24))) << 24);
        __1.z=__1.z & ~(0x000000ff << 0) |((((char)(v_.z >> 0))+((char)(v__1.z >> 0))) << 0);
        __1.z=__1.z & ~(0x000000ff << 8) |((((char)(v_.z >> 8))+((char)(v__1.z >> 8))) << 8);
        __1.z=__1.z & ~(0x000000ff << 16) |((((char)(v_.z >> 16))+((char)(v__1.z >> 16))) << 16);
        __1.z=__1.z & ~(0x000000ff << 24) |((((char)(v_.z >> 24))+((char)(v__1.z >> 24))) << 24);
        __1.w=__1.w & ~(0x000000ff << 0) |((((char)(v_.w >> 0))+((char)(v__1.w >> 0))) << 0);
        __1.w=__1.w & ~(0x000000ff << 8) |((((char)(v_.w >> 8))+((char)(v__1.w >> 8))) << 8);
        __1.w=__1.w & ~(0x000000ff << 16) |((((char)(v_.w >> 16))+((char)(v__1.w >> 16))) << 16);
        __1.w=__1.w & ~(0x000000ff << 24) |((((char)(v_.w >> 24))+((char)(v__1.w >> 24))) << 24);
      *(int4*)(T_add_n_513_to_768_ + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)767) / (int64_t)768) + (((n + (int64_t)767) % (int64_t)768) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_769_to_1024__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_769_to_1024_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < ((n + (int64_t)1023) >> (int64_t)10); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * ((n + (int64_t)1023) >> (int64_t)10)) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)160)) < (int64_t)0) {
      int4 __1;
        int4 v_ = *(int4*)(A + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * ((n + (int64_t)1023) >> (int64_t)10)) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        int4 v__1 = *(int4*)(B + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * ((n + (int64_t)1023) >> (int64_t)10)) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        __1.x=((((char)(v_.x >> 0))+((char)(v__1.x >> 0))) << 0);
        __1.x=__1.x & ~(0x000000ff << 8) |((((char)(v_.x >> 8))+((char)(v__1.x >> 8))) << 8);
        __1.x=__1.x & ~(0x000000ff << 16) |((((char)(v_.x >> 16))+((char)(v__1.x >> 16))) << 16);
        __1.x=__1.x & ~(0x000000ff << 24) |((((char)(v_.x >> 24))+((char)(v__1.x >> 24))) << 24);
        __1.y=__1.y & ~(0x000000ff << 0) |((((char)(v_.y >> 0))+((char)(v__1.y >> 0))) << 0);
        __1.y=__1.y & ~(0x000000ff << 8) |((((char)(v_.y >> 8))+((char)(v__1.y >> 8))) << 8);
        __1.y=__1.y & ~(0x000000ff << 16) |((((char)(v_.y >> 16))+((char)(v__1.y >> 16))) << 16);
        __1.y=__1.y & ~(0x000000ff << 24) |((((char)(v_.y >> 24))+((char)(v__1.y >> 24))) << 24);
        __1.z=__1.z & ~(0x000000ff << 0) |((((char)(v_.z >> 0))+((char)(v__1.z >> 0))) << 0);
        __1.z=__1.z & ~(0x000000ff << 8) |((((char)(v_.z >> 8))+((char)(v__1.z >> 8))) << 8);
        __1.z=__1.z & ~(0x000000ff << 16) |((((char)(v_.z >> 16))+((char)(v__1.z >> 16))) << 16);
        __1.z=__1.z & ~(0x000000ff << 24) |((((char)(v_.z >> 24))+((char)(v__1.z >> 24))) << 24);
        __1.w=__1.w & ~(0x000000ff << 0) |((((char)(v_.w >> 0))+((char)(v__1.w >> 0))) << 0);
        __1.w=__1.w & ~(0x000000ff << 8) |((((char)(v_.w >> 8))+((char)(v__1.w >> 8))) << 8);
        __1.w=__1.w & ~(0x000000ff << 16) |((((char)(v_.w >> 16))+((char)(v__1.w >> 16))) << 16);
        __1.w=__1.w & ~(0x000000ff << 24) |((((char)(v_.w >> 24))+((char)(v__1.w >> 24))) << 24);
      *(int4*)(T_add_n_769_to_1024_ + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * ((n + (int64_t)1023) >> (int64_t)10)) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_1025_to_1280__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_1025_to_1280_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)1279) / (int64_t)1280) + (((n + (int64_t)1279) % (int64_t)1280) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)1279) / (int64_t)1280) + (((n + (int64_t)1279) % (int64_t)1280) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)160)) < (int64_t)0) {
      int4 __1;
        int4 v_ = *(int4*)(A + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1279) / (int64_t)1280) + (((n + (int64_t)1279) % (int64_t)1280) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        int4 v__1 = *(int4*)(B + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1279) / (int64_t)1280) + (((n + (int64_t)1279) % (int64_t)1280) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        __1.x=((((char)(v_.x >> 0))+((char)(v__1.x >> 0))) << 0);
        __1.x=__1.x & ~(0x000000ff << 8) |((((char)(v_.x >> 8))+((char)(v__1.x >> 8))) << 8);
        __1.x=__1.x & ~(0x000000ff << 16) |((((char)(v_.x >> 16))+((char)(v__1.x >> 16))) << 16);
        __1.x=__1.x & ~(0x000000ff << 24) |((((char)(v_.x >> 24))+((char)(v__1.x >> 24))) << 24);
        __1.y=__1.y & ~(0x000000ff << 0) |((((char)(v_.y >> 0))+((char)(v__1.y >> 0))) << 0);
        __1.y=__1.y & ~(0x000000ff << 8) |((((char)(v_.y >> 8))+((char)(v__1.y >> 8))) << 8);
        __1.y=__1.y & ~(0x000000ff << 16) |((((char)(v_.y >> 16))+((char)(v__1.y >> 16))) << 16);
        __1.y=__1.y & ~(0x000000ff << 24) |((((char)(v_.y >> 24))+((char)(v__1.y >> 24))) << 24);
        __1.z=__1.z & ~(0x000000ff << 0) |((((char)(v_.z >> 0))+((char)(v__1.z >> 0))) << 0);
        __1.z=__1.z & ~(0x000000ff << 8) |((((char)(v_.z >> 8))+((char)(v__1.z >> 8))) << 8);
        __1.z=__1.z & ~(0x000000ff << 16) |((((char)(v_.z >> 16))+((char)(v__1.z >> 16))) << 16);
        __1.z=__1.z & ~(0x000000ff << 24) |((((char)(v_.z >> 24))+((char)(v__1.z >> 24))) << 24);
        __1.w=__1.w & ~(0x000000ff << 0) |((((char)(v_.w >> 0))+((char)(v__1.w >> 0))) << 0);
        __1.w=__1.w & ~(0x000000ff << 8) |((((char)(v_.w >> 8))+((char)(v__1.w >> 8))) << 8);
        __1.w=__1.w & ~(0x000000ff << 16) |((((char)(v_.w >> 16))+((char)(v__1.w >> 16))) << 16);
        __1.w=__1.w & ~(0x000000ff << 24) |((((char)(v_.w >> 24))+((char)(v__1.w >> 24))) << 24);
      *(int4*)(T_add_n_1025_to_1280_ + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1279) / (int64_t)1280) + (((n + (int64_t)1279) % (int64_t)1280) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_1281_to_1536__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_1281_to_1536_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)1535) / (int64_t)1536) + (((n + (int64_t)1535) % (int64_t)1536) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)1535) / (int64_t)1536) + (((n + (int64_t)1535) % (int64_t)1536) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)160)) < (int64_t)0) {
      int4 __1;
        int4 v_ = *(int4*)(A + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1535) / (int64_t)1536) + (((n + (int64_t)1535) % (int64_t)1536) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        int4 v__1 = *(int4*)(B + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1535) / (int64_t)1536) + (((n + (int64_t)1535) % (int64_t)1536) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        __1.x=((((char)(v_.x >> 0))+((char)(v__1.x >> 0))) << 0);
        __1.x=__1.x & ~(0x000000ff << 8) |((((char)(v_.x >> 8))+((char)(v__1.x >> 8))) << 8);
        __1.x=__1.x & ~(0x000000ff << 16) |((((char)(v_.x >> 16))+((char)(v__1.x >> 16))) << 16);
        __1.x=__1.x & ~(0x000000ff << 24) |((((char)(v_.x >> 24))+((char)(v__1.x >> 24))) << 24);
        __1.y=__1.y & ~(0x000000ff << 0) |((((char)(v_.y >> 0))+((char)(v__1.y >> 0))) << 0);
        __1.y=__1.y & ~(0x000000ff << 8) |((((char)(v_.y >> 8))+((char)(v__1.y >> 8))) << 8);
        __1.y=__1.y & ~(0x000000ff << 16) |((((char)(v_.y >> 16))+((char)(v__1.y >> 16))) << 16);
        __1.y=__1.y & ~(0x000000ff << 24) |((((char)(v_.y >> 24))+((char)(v__1.y >> 24))) << 24);
        __1.z=__1.z & ~(0x000000ff << 0) |((((char)(v_.z >> 0))+((char)(v__1.z >> 0))) << 0);
        __1.z=__1.z & ~(0x000000ff << 8) |((((char)(v_.z >> 8))+((char)(v__1.z >> 8))) << 8);
        __1.z=__1.z & ~(0x000000ff << 16) |((((char)(v_.z >> 16))+((char)(v__1.z >> 16))) << 16);
        __1.z=__1.z & ~(0x000000ff << 24) |((((char)(v_.z >> 24))+((char)(v__1.z >> 24))) << 24);
        __1.w=__1.w & ~(0x000000ff << 0) |((((char)(v_.w >> 0))+((char)(v__1.w >> 0))) << 0);
        __1.w=__1.w & ~(0x000000ff << 8) |((((char)(v_.w >> 8))+((char)(v__1.w >> 8))) << 8);
        __1.w=__1.w & ~(0x000000ff << 16) |((((char)(v_.w >> 16))+((char)(v__1.w >> 16))) << 16);
        __1.w=__1.w & ~(0x000000ff << 24) |((((char)(v_.w >> 24))+((char)(v__1.w >> 24))) << 24);
      *(int4*)(T_add_n_1281_to_1536_ + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1535) / (int64_t)1536) + (((n + (int64_t)1535) % (int64_t)1536) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_1537_to_1792__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_1537_to_1792_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)1791) / (int64_t)1792) + (((n + (int64_t)1791) % (int64_t)1792) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)1791) / (int64_t)1792) + (((n + (int64_t)1791) % (int64_t)1792) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)160)) < (int64_t)0) {
      int4 __1;
        int4 v_ = *(int4*)(A + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1791) / (int64_t)1792) + (((n + (int64_t)1791) % (int64_t)1792) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        int4 v__1 = *(int4*)(B + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1791) / (int64_t)1792) + (((n + (int64_t)1791) % (int64_t)1792) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        __1.x=((((char)(v_.x >> 0))+((char)(v__1.x >> 0))) << 0);
        __1.x=__1.x & ~(0x000000ff << 8) |((((char)(v_.x >> 8))+((char)(v__1.x >> 8))) << 8);
        __1.x=__1.x & ~(0x000000ff << 16) |((((char)(v_.x >> 16))+((char)(v__1.x >> 16))) << 16);
        __1.x=__1.x & ~(0x000000ff << 24) |((((char)(v_.x >> 24))+((char)(v__1.x >> 24))) << 24);
        __1.y=__1.y & ~(0x000000ff << 0) |((((char)(v_.y >> 0))+((char)(v__1.y >> 0))) << 0);
        __1.y=__1.y & ~(0x000000ff << 8) |((((char)(v_.y >> 8))+((char)(v__1.y >> 8))) << 8);
        __1.y=__1.y & ~(0x000000ff << 16) |((((char)(v_.y >> 16))+((char)(v__1.y >> 16))) << 16);
        __1.y=__1.y & ~(0x000000ff << 24) |((((char)(v_.y >> 24))+((char)(v__1.y >> 24))) << 24);
        __1.z=__1.z & ~(0x000000ff << 0) |((((char)(v_.z >> 0))+((char)(v__1.z >> 0))) << 0);
        __1.z=__1.z & ~(0x000000ff << 8) |((((char)(v_.z >> 8))+((char)(v__1.z >> 8))) << 8);
        __1.z=__1.z & ~(0x000000ff << 16) |((((char)(v_.z >> 16))+((char)(v__1.z >> 16))) << 16);
        __1.z=__1.z & ~(0x000000ff << 24) |((((char)(v_.z >> 24))+((char)(v__1.z >> 24))) << 24);
        __1.w=__1.w & ~(0x000000ff << 0) |((((char)(v_.w >> 0))+((char)(v__1.w >> 0))) << 0);
        __1.w=__1.w & ~(0x000000ff << 8) |((((char)(v_.w >> 8))+((char)(v__1.w >> 8))) << 8);
        __1.w=__1.w & ~(0x000000ff << 16) |((((char)(v_.w >> 16))+((char)(v__1.w >> 16))) << 16);
        __1.w=__1.w & ~(0x000000ff << 24) |((((char)(v_.w >> 24))+((char)(v__1.w >> 24))) << 24);
      *(int4*)(T_add_n_1537_to_1792_ + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)1791) / (int64_t)1792) + (((n + (int64_t)1791) % (int64_t)1792) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_1793_to_2048__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_1793_to_2048_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < ((n + (int64_t)2047) >> (int64_t)11); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * ((n + (int64_t)2047) >> (int64_t)11)) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)160)) < (int64_t)0) {
      int4 __1;
        int4 v_ = *(int4*)(A + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * ((n + (int64_t)2047) >> (int64_t)11)) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        int4 v__1 = *(int4*)(B + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * ((n + (int64_t)2047) >> (int64_t)11)) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        __1.x=((((char)(v_.x >> 0))+((char)(v__1.x >> 0))) << 0);
        __1.x=__1.x & ~(0x000000ff << 8) |((((char)(v_.x >> 8))+((char)(v__1.x >> 8))) << 8);
        __1.x=__1.x & ~(0x000000ff << 16) |((((char)(v_.x >> 16))+((char)(v__1.x >> 16))) << 16);
        __1.x=__1.x & ~(0x000000ff << 24) |((((char)(v_.x >> 24))+((char)(v__1.x >> 24))) << 24);
        __1.y=__1.y & ~(0x000000ff << 0) |((((char)(v_.y >> 0))+((char)(v__1.y >> 0))) << 0);
        __1.y=__1.y & ~(0x000000ff << 8) |((((char)(v_.y >> 8))+((char)(v__1.y >> 8))) << 8);
        __1.y=__1.y & ~(0x000000ff << 16) |((((char)(v_.y >> 16))+((char)(v__1.y >> 16))) << 16);
        __1.y=__1.y & ~(0x000000ff << 24) |((((char)(v_.y >> 24))+((char)(v__1.y >> 24))) << 24);
        __1.z=__1.z & ~(0x000000ff << 0) |((((char)(v_.z >> 0))+((char)(v__1.z >> 0))) << 0);
        __1.z=__1.z & ~(0x000000ff << 8) |((((char)(v_.z >> 8))+((char)(v__1.z >> 8))) << 8);
        __1.z=__1.z & ~(0x000000ff << 16) |((((char)(v_.z >> 16))+((char)(v__1.z >> 16))) << 16);
        __1.z=__1.z & ~(0x000000ff << 24) |((((char)(v_.z >> 24))+((char)(v__1.z >> 24))) << 24);
        __1.w=__1.w & ~(0x000000ff << 0) |((((char)(v_.w >> 0))+((char)(v__1.w >> 0))) << 0);
        __1.w=__1.w & ~(0x000000ff << 8) |((((char)(v_.w >> 8))+((char)(v__1.w >> 8))) << 8);
        __1.w=__1.w & ~(0x000000ff << 16) |((((char)(v_.w >> 16))+((char)(v__1.w >> 16))) << 16);
        __1.w=__1.w & ~(0x000000ff << 24) |((((char)(v_.w >> 24))+((char)(v__1.w >> 24))) << 24);
      *(int4*)(T_add_n_1793_to_2048_ + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * ((n + (int64_t)2047) >> (int64_t)11)) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_2049_to_2304__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_2049_to_2304_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)2303) / (int64_t)2304) + (((n + (int64_t)2303) % (int64_t)2304) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)2303) / (int64_t)2304) + (((n + (int64_t)2303) % (int64_t)2304) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)160)) < (int64_t)0) {
      int4 __1;
        int4 v_ = *(int4*)(A + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2303) / (int64_t)2304) + (((n + (int64_t)2303) % (int64_t)2304) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        int4 v__1 = *(int4*)(B + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2303) / (int64_t)2304) + (((n + (int64_t)2303) % (int64_t)2304) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        __1.x=((((char)(v_.x >> 0))+((char)(v__1.x >> 0))) << 0);
        __1.x=__1.x & ~(0x000000ff << 8) |((((char)(v_.x >> 8))+((char)(v__1.x >> 8))) << 8);
        __1.x=__1.x & ~(0x000000ff << 16) |((((char)(v_.x >> 16))+((char)(v__1.x >> 16))) << 16);
        __1.x=__1.x & ~(0x000000ff << 24) |((((char)(v_.x >> 24))+((char)(v__1.x >> 24))) << 24);
        __1.y=__1.y & ~(0x000000ff << 0) |((((char)(v_.y >> 0))+((char)(v__1.y >> 0))) << 0);
        __1.y=__1.y & ~(0x000000ff << 8) |((((char)(v_.y >> 8))+((char)(v__1.y >> 8))) << 8);
        __1.y=__1.y & ~(0x000000ff << 16) |((((char)(v_.y >> 16))+((char)(v__1.y >> 16))) << 16);
        __1.y=__1.y & ~(0x000000ff << 24) |((((char)(v_.y >> 24))+((char)(v__1.y >> 24))) << 24);
        __1.z=__1.z & ~(0x000000ff << 0) |((((char)(v_.z >> 0))+((char)(v__1.z >> 0))) << 0);
        __1.z=__1.z & ~(0x000000ff << 8) |((((char)(v_.z >> 8))+((char)(v__1.z >> 8))) << 8);
        __1.z=__1.z & ~(0x000000ff << 16) |((((char)(v_.z >> 16))+((char)(v__1.z >> 16))) << 16);
        __1.z=__1.z & ~(0x000000ff << 24) |((((char)(v_.z >> 24))+((char)(v__1.z >> 24))) << 24);
        __1.w=__1.w & ~(0x000000ff << 0) |((((char)(v_.w >> 0))+((char)(v__1.w >> 0))) << 0);
        __1.w=__1.w & ~(0x000000ff << 8) |((((char)(v_.w >> 8))+((char)(v__1.w >> 8))) << 8);
        __1.w=__1.w & ~(0x000000ff << 16) |((((char)(v_.w >> 16))+((char)(v__1.w >> 16))) << 16);
        __1.w=__1.w & ~(0x000000ff << 24) |((((char)(v_.w >> 24))+((char)(v__1.w >> 24))) << 24);
      *(int4*)(T_add_n_2049_to_2304_ + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2303) / (int64_t)2304) + (((n + (int64_t)2303) % (int64_t)2304) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_2305_to_2560__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_2305_to_2560_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)2559) / (int64_t)2560) + (((n + (int64_t)2559) % (int64_t)2560) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)2559) / (int64_t)2560) + (((n + (int64_t)2559) % (int64_t)2560) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)160)) < (int64_t)0) {
      int4 __1;
        int4 v_ = *(int4*)(A + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2559) / (int64_t)2560) + (((n + (int64_t)2559) % (int64_t)2560) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        int4 v__1 = *(int4*)(B + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2559) / (int64_t)2560) + (((n + (int64_t)2559) % (int64_t)2560) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        __1.x=((((char)(v_.x >> 0))+((char)(v__1.x >> 0))) << 0);
        __1.x=__1.x & ~(0x000000ff << 8) |((((char)(v_.x >> 8))+((char)(v__1.x >> 8))) << 8);
        __1.x=__1.x & ~(0x000000ff << 16) |((((char)(v_.x >> 16))+((char)(v__1.x >> 16))) << 16);
        __1.x=__1.x & ~(0x000000ff << 24) |((((char)(v_.x >> 24))+((char)(v__1.x >> 24))) << 24);
        __1.y=__1.y & ~(0x000000ff << 0) |((((char)(v_.y >> 0))+((char)(v__1.y >> 0))) << 0);
        __1.y=__1.y & ~(0x000000ff << 8) |((((char)(v_.y >> 8))+((char)(v__1.y >> 8))) << 8);
        __1.y=__1.y & ~(0x000000ff << 16) |((((char)(v_.y >> 16))+((char)(v__1.y >> 16))) << 16);
        __1.y=__1.y & ~(0x000000ff << 24) |((((char)(v_.y >> 24))+((char)(v__1.y >> 24))) << 24);
        __1.z=__1.z & ~(0x000000ff << 0) |((((char)(v_.z >> 0))+((char)(v__1.z >> 0))) << 0);
        __1.z=__1.z & ~(0x000000ff << 8) |((((char)(v_.z >> 8))+((char)(v__1.z >> 8))) << 8);
        __1.z=__1.z & ~(0x000000ff << 16) |((((char)(v_.z >> 16))+((char)(v__1.z >> 16))) << 16);
        __1.z=__1.z & ~(0x000000ff << 24) |((((char)(v_.z >> 24))+((char)(v__1.z >> 24))) << 24);
        __1.w=__1.w & ~(0x000000ff << 0) |((((char)(v_.w >> 0))+((char)(v__1.w >> 0))) << 0);
        __1.w=__1.w & ~(0x000000ff << 8) |((((char)(v_.w >> 8))+((char)(v__1.w >> 8))) << 8);
        __1.w=__1.w & ~(0x000000ff << 16) |((((char)(v_.w >> 16))+((char)(v__1.w >> 16))) << 16);
        __1.w=__1.w & ~(0x000000ff << 24) |((((char)(v_.w >> 24))+((char)(v__1.w >> 24))) << 24);
      *(int4*)(T_add_n_2305_to_2560_ + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2559) / (int64_t)2560) + (((n + (int64_t)2559) % (int64_t)2560) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_2561_to_2816__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_2561_to_2816_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)2815) / (int64_t)2816) + (((n + (int64_t)2815) % (int64_t)2816) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)2815) / (int64_t)2816) + (((n + (int64_t)2815) % (int64_t)2816) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)160)) < (int64_t)0) {
      int4 __1;
        int4 v_ = *(int4*)(A + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2815) / (int64_t)2816) + (((n + (int64_t)2815) % (int64_t)2816) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        int4 v__1 = *(int4*)(B + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2815) / (int64_t)2816) + (((n + (int64_t)2815) % (int64_t)2816) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        __1.x=((((char)(v_.x >> 0))+((char)(v__1.x >> 0))) << 0);
        __1.x=__1.x & ~(0x000000ff << 8) |((((char)(v_.x >> 8))+((char)(v__1.x >> 8))) << 8);
        __1.x=__1.x & ~(0x000000ff << 16) |((((char)(v_.x >> 16))+((char)(v__1.x >> 16))) << 16);
        __1.x=__1.x & ~(0x000000ff << 24) |((((char)(v_.x >> 24))+((char)(v__1.x >> 24))) << 24);
        __1.y=__1.y & ~(0x000000ff << 0) |((((char)(v_.y >> 0))+((char)(v__1.y >> 0))) << 0);
        __1.y=__1.y & ~(0x000000ff << 8) |((((char)(v_.y >> 8))+((char)(v__1.y >> 8))) << 8);
        __1.y=__1.y & ~(0x000000ff << 16) |((((char)(v_.y >> 16))+((char)(v__1.y >> 16))) << 16);
        __1.y=__1.y & ~(0x000000ff << 24) |((((char)(v_.y >> 24))+((char)(v__1.y >> 24))) << 24);
        __1.z=__1.z & ~(0x000000ff << 0) |((((char)(v_.z >> 0))+((char)(v__1.z >> 0))) << 0);
        __1.z=__1.z & ~(0x000000ff << 8) |((((char)(v_.z >> 8))+((char)(v__1.z >> 8))) << 8);
        __1.z=__1.z & ~(0x000000ff << 16) |((((char)(v_.z >> 16))+((char)(v__1.z >> 16))) << 16);
        __1.z=__1.z & ~(0x000000ff << 24) |((((char)(v_.z >> 24))+((char)(v__1.z >> 24))) << 24);
        __1.w=__1.w & ~(0x000000ff << 0) |((((char)(v_.w >> 0))+((char)(v__1.w >> 0))) << 0);
        __1.w=__1.w & ~(0x000000ff << 8) |((((char)(v_.w >> 8))+((char)(v__1.w >> 8))) << 8);
        __1.w=__1.w & ~(0x000000ff << 16) |((((char)(v_.w >> 16))+((char)(v__1.w >> 16))) << 16);
        __1.w=__1.w & ~(0x000000ff << 24) |((((char)(v_.w >> 24))+((char)(v__1.w >> 24))) << 24);
      *(int4*)(T_add_n_2561_to_2816_ + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)2815) / (int64_t)2816) + (((n + (int64_t)2815) % (int64_t)2816) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_2817_to_3072__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_2817_to_3072_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)3071) / (int64_t)3072) + (((n + (int64_t)3071) % (int64_t)3072) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)3071) / (int64_t)3072) + (((n + (int64_t)3071) % (int64_t)3072) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)160)) < (int64_t)0) {
      int4 __1;
        int4 v_ = *(int4*)(A + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3071) / (int64_t)3072) + (((n + (int64_t)3071) % (int64_t)3072) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        int4 v__1 = *(int4*)(B + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3071) / (int64_t)3072) + (((n + (int64_t)3071) % (int64_t)3072) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        __1.x=((((char)(v_.x >> 0))+((char)(v__1.x >> 0))) << 0);
        __1.x=__1.x & ~(0x000000ff << 8) |((((char)(v_.x >> 8))+((char)(v__1.x >> 8))) << 8);
        __1.x=__1.x & ~(0x000000ff << 16) |((((char)(v_.x >> 16))+((char)(v__1.x >> 16))) << 16);
        __1.x=__1.x & ~(0x000000ff << 24) |((((char)(v_.x >> 24))+((char)(v__1.x >> 24))) << 24);
        __1.y=__1.y & ~(0x000000ff << 0) |((((char)(v_.y >> 0))+((char)(v__1.y >> 0))) << 0);
        __1.y=__1.y & ~(0x000000ff << 8) |((((char)(v_.y >> 8))+((char)(v__1.y >> 8))) << 8);
        __1.y=__1.y & ~(0x000000ff << 16) |((((char)(v_.y >> 16))+((char)(v__1.y >> 16))) << 16);
        __1.y=__1.y & ~(0x000000ff << 24) |((((char)(v_.y >> 24))+((char)(v__1.y >> 24))) << 24);
        __1.z=__1.z & ~(0x000000ff << 0) |((((char)(v_.z >> 0))+((char)(v__1.z >> 0))) << 0);
        __1.z=__1.z & ~(0x000000ff << 8) |((((char)(v_.z >> 8))+((char)(v__1.z >> 8))) << 8);
        __1.z=__1.z & ~(0x000000ff << 16) |((((char)(v_.z >> 16))+((char)(v__1.z >> 16))) << 16);
        __1.z=__1.z & ~(0x000000ff << 24) |((((char)(v_.z >> 24))+((char)(v__1.z >> 24))) << 24);
        __1.w=__1.w & ~(0x000000ff << 0) |((((char)(v_.w >> 0))+((char)(v__1.w >> 0))) << 0);
        __1.w=__1.w & ~(0x000000ff << 8) |((((char)(v_.w >> 8))+((char)(v__1.w >> 8))) << 8);
        __1.w=__1.w & ~(0x000000ff << 16) |((((char)(v_.w >> 16))+((char)(v__1.w >> 16))) << 16);
        __1.w=__1.w & ~(0x000000ff << 24) |((((char)(v_.w >> 24))+((char)(v__1.w >> 24))) << 24);
      *(int4*)(T_add_n_2817_to_3072_ + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3071) / (int64_t)3072) + (((n + (int64_t)3071) % (int64_t)3072) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_3073_to_3328__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_3073_to_3328_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)3327) / (int64_t)3328) + (((n + (int64_t)3327) % (int64_t)3328) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)3327) / (int64_t)3328) + (((n + (int64_t)3327) % (int64_t)3328) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)160)) < (int64_t)0) {
      int4 __1;
        int4 v_ = *(int4*)(A + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3327) / (int64_t)3328) + (((n + (int64_t)3327) % (int64_t)3328) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        int4 v__1 = *(int4*)(B + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3327) / (int64_t)3328) + (((n + (int64_t)3327) % (int64_t)3328) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        __1.x=((((char)(v_.x >> 0))+((char)(v__1.x >> 0))) << 0);
        __1.x=__1.x & ~(0x000000ff << 8) |((((char)(v_.x >> 8))+((char)(v__1.x >> 8))) << 8);
        __1.x=__1.x & ~(0x000000ff << 16) |((((char)(v_.x >> 16))+((char)(v__1.x >> 16))) << 16);
        __1.x=__1.x & ~(0x000000ff << 24) |((((char)(v_.x >> 24))+((char)(v__1.x >> 24))) << 24);
        __1.y=__1.y & ~(0x000000ff << 0) |((((char)(v_.y >> 0))+((char)(v__1.y >> 0))) << 0);
        __1.y=__1.y & ~(0x000000ff << 8) |((((char)(v_.y >> 8))+((char)(v__1.y >> 8))) << 8);
        __1.y=__1.y & ~(0x000000ff << 16) |((((char)(v_.y >> 16))+((char)(v__1.y >> 16))) << 16);
        __1.y=__1.y & ~(0x000000ff << 24) |((((char)(v_.y >> 24))+((char)(v__1.y >> 24))) << 24);
        __1.z=__1.z & ~(0x000000ff << 0) |((((char)(v_.z >> 0))+((char)(v__1.z >> 0))) << 0);
        __1.z=__1.z & ~(0x000000ff << 8) |((((char)(v_.z >> 8))+((char)(v__1.z >> 8))) << 8);
        __1.z=__1.z & ~(0x000000ff << 16) |((((char)(v_.z >> 16))+((char)(v__1.z >> 16))) << 16);
        __1.z=__1.z & ~(0x000000ff << 24) |((((char)(v_.z >> 24))+((char)(v__1.z >> 24))) << 24);
        __1.w=__1.w & ~(0x000000ff << 0) |((((char)(v_.w >> 0))+((char)(v__1.w >> 0))) << 0);
        __1.w=__1.w & ~(0x000000ff << 8) |((((char)(v_.w >> 8))+((char)(v__1.w >> 8))) << 8);
        __1.w=__1.w & ~(0x000000ff << 16) |((((char)(v_.w >> 16))+((char)(v__1.w >> 16))) << 16);
        __1.w=__1.w & ~(0x000000ff << 24) |((((char)(v_.w >> 24))+((char)(v__1.w >> 24))) << 24);
      *(int4*)(T_add_n_3073_to_3328_ + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3327) / (int64_t)3328) + (((n + (int64_t)3327) % (int64_t)3328) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_3329_to_3584__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_3329_to_3584_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)3583) / (int64_t)3584) + (((n + (int64_t)3583) % (int64_t)3584) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)3583) / (int64_t)3584) + (((n + (int64_t)3583) % (int64_t)3584) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)160)) < (int64_t)0) {
      int4 __1;
        int4 v_ = *(int4*)(A + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3583) / (int64_t)3584) + (((n + (int64_t)3583) % (int64_t)3584) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        int4 v__1 = *(int4*)(B + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3583) / (int64_t)3584) + (((n + (int64_t)3583) % (int64_t)3584) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        __1.x=((((char)(v_.x >> 0))+((char)(v__1.x >> 0))) << 0);
        __1.x=__1.x & ~(0x000000ff << 8) |((((char)(v_.x >> 8))+((char)(v__1.x >> 8))) << 8);
        __1.x=__1.x & ~(0x000000ff << 16) |((((char)(v_.x >> 16))+((char)(v__1.x >> 16))) << 16);
        __1.x=__1.x & ~(0x000000ff << 24) |((((char)(v_.x >> 24))+((char)(v__1.x >> 24))) << 24);
        __1.y=__1.y & ~(0x000000ff << 0) |((((char)(v_.y >> 0))+((char)(v__1.y >> 0))) << 0);
        __1.y=__1.y & ~(0x000000ff << 8) |((((char)(v_.y >> 8))+((char)(v__1.y >> 8))) << 8);
        __1.y=__1.y & ~(0x000000ff << 16) |((((char)(v_.y >> 16))+((char)(v__1.y >> 16))) << 16);
        __1.y=__1.y & ~(0x000000ff << 24) |((((char)(v_.y >> 24))+((char)(v__1.y >> 24))) << 24);
        __1.z=__1.z & ~(0x000000ff << 0) |((((char)(v_.z >> 0))+((char)(v__1.z >> 0))) << 0);
        __1.z=__1.z & ~(0x000000ff << 8) |((((char)(v_.z >> 8))+((char)(v__1.z >> 8))) << 8);
        __1.z=__1.z & ~(0x000000ff << 16) |((((char)(v_.z >> 16))+((char)(v__1.z >> 16))) << 16);
        __1.z=__1.z & ~(0x000000ff << 24) |((((char)(v_.z >> 24))+((char)(v__1.z >> 24))) << 24);
        __1.w=__1.w & ~(0x000000ff << 0) |((((char)(v_.w >> 0))+((char)(v__1.w >> 0))) << 0);
        __1.w=__1.w & ~(0x000000ff << 8) |((((char)(v_.w >> 8))+((char)(v__1.w >> 8))) << 8);
        __1.w=__1.w & ~(0x000000ff << 16) |((((char)(v_.w >> 16))+((char)(v__1.w >> 16))) << 16);
        __1.w=__1.w & ~(0x000000ff << 24) |((((char)(v_.w >> 24))+((char)(v__1.w >> 24))) << 24);
      *(int4*)(T_add_n_3329_to_3584_ + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3583) / (int64_t)3584) + (((n + (int64_t)3583) % (int64_t)3584) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_3585_to_3840__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_3585_to_3840_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < (((n + (int64_t)3839) / (int64_t)3840) + (((n + (int64_t)3839) % (int64_t)3840) >> (int64_t)63)); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * (((n + (int64_t)3839) / (int64_t)3840) + (((n + (int64_t)3839) % (int64_t)3840) >> (int64_t)63))) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)160)) < (int64_t)0) {
      int4 __1;
        int4 v_ = *(int4*)(A + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3839) / (int64_t)3840) + (((n + (int64_t)3839) % (int64_t)3840) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        int4 v__1 = *(int4*)(B + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3839) / (int64_t)3840) + (((n + (int64_t)3839) % (int64_t)3840) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        __1.x=((((char)(v_.x >> 0))+((char)(v__1.x >> 0))) << 0);
        __1.x=__1.x & ~(0x000000ff << 8) |((((char)(v_.x >> 8))+((char)(v__1.x >> 8))) << 8);
        __1.x=__1.x & ~(0x000000ff << 16) |((((char)(v_.x >> 16))+((char)(v__1.x >> 16))) << 16);
        __1.x=__1.x & ~(0x000000ff << 24) |((((char)(v_.x >> 24))+((char)(v__1.x >> 24))) << 24);
        __1.y=__1.y & ~(0x000000ff << 0) |((((char)(v_.y >> 0))+((char)(v__1.y >> 0))) << 0);
        __1.y=__1.y & ~(0x000000ff << 8) |((((char)(v_.y >> 8))+((char)(v__1.y >> 8))) << 8);
        __1.y=__1.y & ~(0x000000ff << 16) |((((char)(v_.y >> 16))+((char)(v__1.y >> 16))) << 16);
        __1.y=__1.y & ~(0x000000ff << 24) |((((char)(v_.y >> 24))+((char)(v__1.y >> 24))) << 24);
        __1.z=__1.z & ~(0x000000ff << 0) |((((char)(v_.z >> 0))+((char)(v__1.z >> 0))) << 0);
        __1.z=__1.z & ~(0x000000ff << 8) |((((char)(v_.z >> 8))+((char)(v__1.z >> 8))) << 8);
        __1.z=__1.z & ~(0x000000ff << 16) |((((char)(v_.z >> 16))+((char)(v__1.z >> 16))) << 16);
        __1.z=__1.z & ~(0x000000ff << 24) |((((char)(v_.z >> 24))+((char)(v__1.z >> 24))) << 24);
        __1.w=__1.w & ~(0x000000ff << 0) |((((char)(v_.w >> 0))+((char)(v__1.w >> 0))) << 0);
        __1.w=__1.w & ~(0x000000ff << 8) |((((char)(v_.w >> 8))+((char)(v__1.w >> 8))) << 8);
        __1.w=__1.w & ~(0x000000ff << 16) |((((char)(v_.w >> 16))+((char)(v__1.w >> 16))) << 16);
        __1.w=__1.w & ~(0x000000ff << 24) |((((char)(v_.w >> 24))+((char)(v__1.w >> 24))) << 24);
      *(int4*)(T_add_n_3585_to_3840_ + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * (((n + (int64_t)3839) / (int64_t)3840) + (((n + (int64_t)3839) % (int64_t)3840) >> (int64_t)63))) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16))) = __1;
    }
  }
}

extern "C" __global__ void __launch_bounds__(256) add_n_3841_to_4096__kernel(signed char* __restrict__ A, signed char* __restrict__ B, signed char* __restrict__ T_add_n_3841_to_4096_, int64_t n) {
  for (int64_t ax0_ax1_fused_1 = 0; ax0_ax1_fused_1 < ((n + (int64_t)4095) >> (int64_t)12); ++ax0_ax1_fused_1) {
    if ((((((((int64_t)blockIdx.x) * ((n + (int64_t)4095) >> (int64_t)12)) * (int64_t)256) + (ax0_ax1_fused_1 * (int64_t)256)) + ((int64_t)threadIdx.x)) - (n * (int64_t)160)) < (int64_t)0) {
      int4 __1;
        int4 v_ = *(int4*)(A + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * ((n + (int64_t)4095) >> (int64_t)12)) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        int4 v__1 = *(int4*)(B + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * ((n + (int64_t)4095) >> (int64_t)12)) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16)));
        __1.x=((((char)(v_.x >> 0))+((char)(v__1.x >> 0))) << 0);
        __1.x=__1.x & ~(0x000000ff << 8) |((((char)(v_.x >> 8))+((char)(v__1.x >> 8))) << 8);
        __1.x=__1.x & ~(0x000000ff << 16) |((((char)(v_.x >> 16))+((char)(v__1.x >> 16))) << 16);
        __1.x=__1.x & ~(0x000000ff << 24) |((((char)(v_.x >> 24))+((char)(v__1.x >> 24))) << 24);
        __1.y=__1.y & ~(0x000000ff << 0) |((((char)(v_.y >> 0))+((char)(v__1.y >> 0))) << 0);
        __1.y=__1.y & ~(0x000000ff << 8) |((((char)(v_.y >> 8))+((char)(v__1.y >> 8))) << 8);
        __1.y=__1.y & ~(0x000000ff << 16) |((((char)(v_.y >> 16))+((char)(v__1.y >> 16))) << 16);
        __1.y=__1.y & ~(0x000000ff << 24) |((((char)(v_.y >> 24))+((char)(v__1.y >> 24))) << 24);
        __1.z=__1.z & ~(0x000000ff << 0) |((((char)(v_.z >> 0))+((char)(v__1.z >> 0))) << 0);
        __1.z=__1.z & ~(0x000000ff << 8) |((((char)(v_.z >> 8))+((char)(v__1.z >> 8))) << 8);
        __1.z=__1.z & ~(0x000000ff << 16) |((((char)(v_.z >> 16))+((char)(v__1.z >> 16))) << 16);
        __1.z=__1.z & ~(0x000000ff << 24) |((((char)(v_.z >> 24))+((char)(v__1.z >> 24))) << 24);
        __1.w=__1.w & ~(0x000000ff << 0) |((((char)(v_.w >> 0))+((char)(v__1.w >> 0))) << 0);
        __1.w=__1.w & ~(0x000000ff << 8) |((((char)(v_.w >> 8))+((char)(v__1.w >> 8))) << 8);
        __1.w=__1.w & ~(0x000000ff << 16) |((((char)(v_.w >> 16))+((char)(v__1.w >> 16))) << 16);
        __1.w=__1.w & ~(0x000000ff << 24) |((((char)(v_.w >> 24))+((char)(v__1.w >> 24))) << 24);
      *(int4*)(T_add_n_3841_to_4096_ + (((ax0_ax1_fused_1 * (int64_t)4096) + ((((int64_t)blockIdx.x) * ((n + (int64_t)4095) >> (int64_t)12)) * (int64_t)4096)) + (((int64_t)threadIdx.x) * (int64_t)16))) = __1;
    }
  }
}

